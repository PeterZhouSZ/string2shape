#include "hip/hip_runtime.h"
#include "pch.h"
#include "GraphToWFObject.h"

#include "Algebra.h"
#include "WFObjUtils.h"
#include "PartOrientationUtils.h"

#include <thrust/reduce.h>

__host__ WFObject WFObjectGenerator::operator()(
	//example shapes
	WFObject & aObj1,
	WFObject & aObj2,
	//example shape graphs
	Graph & aGraph1,
	Graph & aGraph2,
	//target shape graph
	Graph & aGraph3,
	//estimated edge configurations
	thrust::host_vector<unsigned int>& aEdgeTypes1,
	thrust::host_vector<unsigned int>& aEdgeTypes2,
	thrust::host_vector<unsigned int>& aEdgeTypes3)
{

	mOrientations1.init(aObj1, aGraph1);
	mOrientations2.init(aObj2, aGraph2);
	
	thrust::host_vector<float> objSizes1;
	ObjectCenterExporter()(aObj1, objCenters1, objSizes1);
	
	thrust::host_vector<float> objSizes2;
	ObjectCenterExporter()(aObj2, objCenters2, objSizes2);


	WFObject outputObj;

	unsigned int numNodes = (unsigned)aGraph3.intervals.size() - 1u;
	thrust::host_vector<unsigned int> visited(numNodes, 0u);
	thrust::host_vector<unsigned int> intervalsHost(aGraph3.intervals);
	thrust::host_vector<unsigned int> adjacencyValsHost(aGraph3.adjacencyVals);

	if (seedNodeId >= (unsigned int)numNodes)
	{
		std::default_random_engine generator(seed);
		std::uniform_int_distribution<unsigned int> distribution(0u, (unsigned int)numNodes - 1u);
		seedNodeId = distribution(generator);
	}

	unsigned int seedEdgeId = aGraph3.neighborsBegin(seedNodeId);
	unsigned int seedEdgeType = aEdgeTypes3[seedEdgeId];

	unsigned int seedNodeObj1 = aGraph1.adjacencyKeys[findCorresponingEdgeId(aGraph1, aEdgeTypes1, seedEdgeType)];
	unsigned int seedNodeObj2 = aGraph2.adjacencyKeys[findCorresponingEdgeId(aGraph2, aEdgeTypes2, seedEdgeType)];

	if (seedNodeObj1 == (unsigned)-1 && seedNodeObj2 == (unsigned)-1)
	{
		std::cerr << "Failed to initialize WFObject creation at node " << seedNodeId << "\n";
		return outputObj;
	}
	if (seedNodeObj1 != (unsigned)-1)
	{
		thrust::host_vector<unsigned int> subgraphFlags1(aObj1.getNumObjects(), 0u);
		subgraphFlags1[seedNodeObj1] = 1u;
		float3 zero = make_float3(0.f, 0.f, 0.f);
		outputObj = insertPieces(outputObj, aObj1, subgraphFlags1, zero, -objCenters1[seedNodeObj1], make_quaternion4f(0.f,0.f,0.f,1.f));
	}
	else
	{
		thrust::host_vector<unsigned int> subgraphFlags2(aObj2.getNumObjects(), 0u);
		subgraphFlags2[seedNodeObj2] = 1u;
		float3 zero = make_float3(0.f, 0.f, 0.f);
		outputObj = insertPieces(outputObj, aObj2, subgraphFlags2, zero, -objCenters2[seedNodeObj2], make_quaternion4f(0.f, 0.f, 0.f, 1.f));
	}

	std::deque<unsigned int> frontier;
	frontier.push_back(seedNodeId);
	visited[seedNodeId] = 1u;

	unsigned int indertedNodeCount = 0u;
	thrust::host_vector<unsigned int> nodeIdMap(aGraph3.numNodes(), 0u);
	nodeIdMap[seedNodeId] = indertedNodeCount++;
	
	while (!frontier.empty())
	{
		const unsigned int nodeId = frontier.front();
		frontier.pop_front();
		
		const unsigned int nodeCount = 1u;
		thrust::host_vector<unsigned int> nodeIds(1, nodeIdMap[nodeId]);

		thrust::host_vector<float3> vertexBufferHost;
		thrust::host_vector<uint2> vtxRanges;

		VertexBufferUnpacker unpackVertices;
		unpackVertices(outputObj, nodeIds, vtxRanges, vertexBufferHost);

		//Use PCA to compute local coordiante system for each object
		thrust::host_vector<float3> translations(1);
		thrust::host_vector<quaternion4f> rotations(1);
		thrust::host_vector<double> tmpCovMatrix(1 * 3 * 3, 0.f);
		thrust::host_vector<double> tmpDiagonalW(1 * 3);
		thrust::host_vector<double> tmpMatrixV(1 * 3 * 3);
		thrust::host_vector<double> tmpVecRV(1 * 3);

		LocalCoordsEstimator estimateT(
			thrust::raw_pointer_cast(vtxRanges.data()),
			thrust::raw_pointer_cast(vertexBufferHost.data()),
			thrust::raw_pointer_cast(tmpCovMatrix.data()),
			thrust::raw_pointer_cast(tmpDiagonalW.data()),
			thrust::raw_pointer_cast(tmpMatrixV.data()),
			thrust::raw_pointer_cast(tmpVecRV.data()),
			thrust::raw_pointer_cast(translations.data()),
			thrust::raw_pointer_cast(rotations.data())
		);

		estimateT(0);

		float3 translationA = translations[0];
		quaternion4f rotationA = rotations[0];

		for (unsigned int nbrId = intervalsHost[nodeId]; nbrId < intervalsHost[nodeId + 1]; ++nbrId)
		{
			const unsigned int neighborId = adjacencyValsHost[nbrId];
			if (visited[neighborId] == 0u)
			{
				frontier.push_back(neighborId);
				visited[neighborId] = 1u;

				unsigned int currentEdgeId = aGraph3.neighborsBegin(nodeId) + nbrId;
				unsigned int currentEdgeType = aEdgeTypes3[currentEdgeId];

				unsigned int correspondingEdgeIdObj1 = findCorresponingEdgeId(aGraph1, aEdgeTypes1, currentEdgeType);

				if (correspondingEdgeIdObj1 == (unsigned)-1)
				{
					std::cerr << "Failed to create WFObject node " << correspondingEdgeIdObj1 << "\n";
					std::cerr << "(After inserting " << indertedNodeCount << " nodes.)\n";

					return outputObj;
				}

				unsigned int correspondingNodeIdObj1 = aGraph1.adjacencyKeys[correspondingEdgeIdObj1];
				
				float3 translationA1 = objCenters1[correspondingNodeIdObj1];
				quaternion4f rotationA1 = mOrientations1.getAbsoluteRotation(correspondingNodeIdObj1);

				quaternion4f relativeR = rotationA * rotationA1.conjugate();
				if (isIdentity(relativeR, 0.001f))
					relativeR = make_quaternion4f(0.f, 0.f, 0.f, 1.f);

				thrust::host_vector<unsigned int> subgraphFlags1(aObj1.getNumObjects(), 0u);
				subgraphFlags1[correspondingNodeIdObj1] = 1u;
				outputObj = insertPieces(outputObj, aObj1, subgraphFlags1, translationA, translationA1, relativeR);

				nodeIdMap[neighborId] = indertedNodeCount++;
			}
		}
	}


	return outputObj;
}

__host__ WFObject WFObjectGenerator::insertPieces(
	const WFObject& aObj1,
	const WFObject& aObj2,
	const thrust::host_vector<unsigned int>& aSubgraphFlags2,
	const float3& aTranslation1,
	const float3& aTranslation2,
	const quaternion4f& aRotation)
{
	thrust::host_vector<unsigned int> subgraphFlags1(aObj1.getNumObjects(), 1u);
	float3 translation1 = aTranslation1;
	float3 translation2 = aTranslation2;
	quaternion4f rotation2 = aRotation;
	WFObject result = WFObjectMerger()(aObj1, translation1, aObj2, translation2, rotation2, subgraphFlags1, aSubgraphFlags2);
	return result;
}

__host__ unsigned int WFObjectGenerator::findCorresponingEdgeId(Graph & aGraph1, thrust::host_vector<unsigned int>& aEdgeTypes1, unsigned int aTargetEdgeType)
{
	std::vector<unsigned int> permutedIds(aEdgeTypes1.size());
	for (unsigned int i = 0u; i < permutedIds.size(); ++i)
		permutedIds[i] = i;
	std::shuffle(permutedIds.begin(), permutedIds.end(), mRNG);

	for (size_t i = 0; i < aEdgeTypes1.size(); ++i)
	{
		size_t edgeId = permutedIds[i];
		if (aEdgeTypes1[edgeId] == aTargetEdgeType)
		{
			return edgeId;
		}
	}
	return (unsigned)-1;
}

__host__ void WFObjectGenerator::translateObj(WFObject & aObj, unsigned int aObjId, const float3 & aTranslation)
{
	thrust::host_vector<unsigned int> processed(aObj.getNumVertices(), 0u);
	for (int faceId = aObj.objects[aObjId].x; faceId < aObj.objects[aObjId].y; ++faceId)
	{
		WFObject::Face face = aObj.faces[faceId];
		size_t vtxId1 = aObj.faces[faceId].vert1;
		size_t vtxId2 = aObj.faces[faceId].vert2;
		size_t vtxId3 = aObj.faces[faceId].vert3;
		if (processed[vtxId1] == 0u)
		{
			processed[vtxId1] = 1u;
			float3 vtx = aObj.vertices[vtxId1];
			aObj.vertices[vtxId1] = vtx + aTranslation;
		}
		if (processed[vtxId2] == 0u)
		{
			processed[vtxId2] = 1u;
			float3 vtx = aObj.vertices[vtxId2];
			aObj.vertices[vtxId2] = vtx + aTranslation;

		}
		if (processed[vtxId3] == 0u)
		{
			processed[vtxId3] = 1u;
			float3 vtx = aObj.vertices[vtxId3];
			aObj.vertices[vtxId3] = vtx + aTranslation;
		}
	}
}
