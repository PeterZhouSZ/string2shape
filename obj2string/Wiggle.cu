#include "hip/hip_runtime.h"
#include "pch.h"
#include "Wiggle.h"

#include <deque>

#include "WFObjUtils.h"
#include "PartOrientationUtils.h"

#include "DebugUtils.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>


__host__ void Wiggle::init(WFObject & aObj, Graph & aGraph)
{
	seed = (unsigned int)std::chrono::system_clock::now().time_since_epoch().count();

	float3 minBound, maxBound;
	ObjectBoundsExporter()(aObj, minBound, maxBound);
	spatialTolerance = std::max(0.01f * len(maxBound - minBound), spatialTolerance);


	//Unpack and upload the vertex buffer
	thrust::host_vector<uint2> vertexRangesHost;
	thrust::host_vector<float3> vertexBufferHost;

	VertexBufferUnpacker unpackVertices;
	unpackVertices(aObj, vertexRangesHost, vertexBufferHost);

	thrust::device_vector<uint2> vertexRangesDevice(vertexRangesHost);
	thrust::device_vector<float3> vertexBufferDevice(vertexBufferHost);


//#ifdef _DEBUG
//	outputDeviceVector("vertex ranges: ", vertexRangesDevice);
//	outputDeviceVector("vertex buffer: ", vertexBufferDevice);
//#endif

	//Use PCA to compute local coordiante system for each object
	thrust::device_vector<float3> outTranslation(aObj.getNumObjects());
	thrust::device_vector<quaternion4f> outRotation(aObj.getNumObjects());

	thrust::device_vector<double> tmpCovMatrix(aObj.getNumObjects() * 3 * 3, 0.f);
	thrust::device_vector<double> tmpDiagonalW(aObj.getNumObjects() * 3);
	thrust::device_vector<double> tmpMatrixV(aObj.getNumObjects() * 3 * 3);
	thrust::device_vector<double> tmpVecRV(aObj.getNumObjects() * 3);

	LocalCoordsEstimator estimateT(
		thrust::raw_pointer_cast(vertexRangesDevice.data()),
		thrust::raw_pointer_cast(vertexBufferDevice.data()),
		thrust::raw_pointer_cast(tmpCovMatrix.data()),
		thrust::raw_pointer_cast(tmpDiagonalW.data()),
		thrust::raw_pointer_cast(tmpMatrixV.data()),
		thrust::raw_pointer_cast(tmpVecRV.data()),
		thrust::raw_pointer_cast(outTranslation.data()),
		thrust::raw_pointer_cast(outRotation.data())
	);

	thrust::counting_iterator<size_t> first(0u);
	thrust::counting_iterator<size_t> last(aObj.getNumObjects());

	thrust::for_each(first, last, estimateT);

//#ifdef _DEBUG
//	outputDeviceVector("translations: ", outTranslation);
//	outputDeviceVector("rotations: ", outRotation);
//#endif

	//Extract and upload node type information
	thrust::host_vector<unsigned int> nodeTypesHost(aGraph.numNodes(), (unsigned int)aObj.materials.size());
	for (size_t nodeId = 0; nodeId < aObj.objects.size(); ++nodeId)
	{
		size_t faceId = aObj.objects[nodeId].x;
		size_t materialId = aObj.faces[faceId].material;
		nodeTypesHost[nodeId] = (unsigned int)materialId;
	}
	thrust::device_vector<unsigned int> nodeTypes(nodeTypesHost);

	thrust::device_vector<unsigned int> neighborTypeKeys(aGraph.numEdges() * 2u);
	thrust::device_vector<unsigned int> neighborTypeVals(aGraph.numEdges() * 2u);
	thrust::device_vector<float3> relativeTranslation(aGraph.numEdges() * 2u);
	thrust::device_vector<quaternion4f> relativeRotation(aGraph.numEdges() * 2u);
	thrust::device_vector<quaternion4f> absoluteRotation(aGraph.numEdges() * 2u);

	TransformationExtractor extractRelativeT(
		nodeTypes.data(),
		neighborTypeKeys.data(),
		neighborTypeVals.data(),
		outTranslation.data(),
		outRotation.data(),
		relativeTranslation.data(),
		relativeRotation.data(),
		absoluteRotation.data()
	);

	thrust::counting_iterator<size_t> lastEdge(aGraph.numEdges() * 2u);

	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(aGraph.adjacencyKeys.begin(), aGraph.adjacencyVals.begin(), first)),
		thrust::make_zip_iterator(thrust::make_tuple(aGraph.adjacencyKeys.end(), aGraph.adjacencyVals.end(), lastEdge)),
		extractRelativeT);


	if(mNeighborTypeKeys.size() == 0u)
	{ 
		//first call of init
		mNeighborTypeKeys = thrust::host_vector<unsigned int>(neighborTypeKeys);
		mNeighborTypeVals = thrust::host_vector<unsigned int>(neighborTypeVals);
		mRelativeTranslation = thrust::host_vector<float3>(relativeTranslation);
		mRelativeRotation = thrust::host_vector<quaternion4f>(relativeRotation);
		mAbsoluteRotation = thrust::host_vector<quaternion4f>(absoluteRotation);
	}
	else
	{
		//init already called, append new data
		size_t oldCount = mNeighborTypeKeys.size();
		mNeighborTypeKeys.resize(oldCount + neighborTypeKeys.size());
		mNeighborTypeVals.resize(oldCount + neighborTypeVals.size());
		mRelativeTranslation.resize(oldCount + relativeTranslation.size());
		mRelativeRotation.resize(oldCount + relativeRotation.size());
		mAbsoluteRotation.resize(oldCount + absoluteRotation.size());

		thrust::copy(neighborTypeKeys.begin(), neighborTypeKeys.end(), mNeighborTypeKeys.begin() + oldCount);
		thrust::copy(neighborTypeVals.begin(), neighborTypeVals.end(), mNeighborTypeVals.begin() + oldCount);
		thrust::copy(relativeTranslation.begin(), relativeTranslation.end(), mRelativeTranslation.begin() + oldCount);
		thrust::copy(relativeRotation.begin(), relativeRotation.end(), mRelativeRotation.begin() + oldCount);
		thrust::copy(absoluteRotation.begin(), absoluteRotation.end(), mAbsoluteRotation.begin() + oldCount);
	}

	//sort by node type
	thrust::sort_by_key(
		mNeighborTypeKeys.begin(),
		mNeighborTypeKeys.end(),
		thrust::make_zip_iterator(thrust::make_tuple(mNeighborTypeVals.begin(), mRelativeTranslation.begin(), mRelativeRotation.begin(), mAbsoluteRotation.begin()))
		);

	//setup search intervals for each node type
	mIntervals.resize(aObj.materials.size() + 1u, 0u);
	for (size_t i = 0u; i < mNeighborTypeKeys.size() - 1u; ++i)
	{
		if (mNeighborTypeKeys[i] < mNeighborTypeKeys[i + 1u])
		{
			mIntervals[mNeighborTypeKeys[i] + 1] = (unsigned)i + 1u;
		}
	}
	//last element
	if (mNeighborTypeKeys.size() > 0u)
		mIntervals[mNeighborTypeKeys[mNeighborTypeKeys.size() - 1u] + 1] = (unsigned)mNeighborTypeKeys.size();

	//fill gaps due to missing node types
	for (size_t i = 1u; i < mIntervals.size(); ++i)
	{
		mIntervals[i] = std::max(mIntervals[i - 1u], mIntervals[i]);
	}

#ifdef _DEBUG
	outputHostVector("translations: ", mRelativeTranslation);
	outputHostVector("rotations: ", mRelativeRotation);
#endif

}

__host__ void Wiggle::fixRelativeTransformations(WFObject & aObj, Graph & aGraph)
{
	numCorrections = 0u;

	size_t numNodes = aObj.objects.size();
	thrust::host_vector<unsigned int> visited(numNodes, 0u);
	thrust::host_vector<unsigned int> intervalsHost(aGraph.intervals);
	thrust::host_vector<unsigned int> adjacencyValsHost(aGraph.adjacencyVals);

	//Extract and upload node type information
	thrust::host_vector<unsigned int> nodeTypesHost(aGraph.numNodes(), (unsigned int)aObj.materials.size());
	for (size_t nodeId = 0; nodeId < aObj.objects.size(); ++nodeId)
	{
		size_t faceId = aObj.objects[nodeId].x;
		size_t materialId = aObj.faces[faceId].material;
		nodeTypesHost[nodeId] = (unsigned int)materialId;
	}


	if (seedNodeId >= (unsigned int)numNodes)
	{
		std::default_random_engine generator(seed);
		std::uniform_int_distribution<unsigned int> distribution(0u, (unsigned int)numNodes - 1u);
		seedNodeId = distribution(generator);
	}
		


	std::deque<unsigned int> frontier;
	frontier.push_back(seedNodeId);
	visited[seedNodeId] = 1u;
	while (!frontier.empty())
	{
		const unsigned int nodeId = frontier.front();
		frontier.pop_front();
		
		processNeighbors(
			aObj,
			nodeId,
			visited,
			intervalsHost,
			adjacencyValsHost,
			nodeTypesHost);

		for (unsigned int nbrId = intervalsHost[nodeId]; nbrId < intervalsHost[nodeId + 1]; ++nbrId)
		{
			const unsigned int nodeId = adjacencyValsHost[nbrId];
			if (visited[nodeId] == 0u)
			{
				frontier.push_back(nodeId);
				visited[nodeId] = 1u;
			}
		}
	}

}

__host__ void Wiggle::processNeighbors(
	WFObject&							aObj,
	unsigned int						aObjId,
	thrust::host_vector<unsigned int>&	visited,
	thrust::host_vector<unsigned int>&	intervalsHost,
	thrust::host_vector<unsigned int>&	adjacencyValsHost,
	thrust::host_vector<unsigned int>&	nodeTypeIds)
{
	const unsigned int nbrCount = intervalsHost[aObjId + 1u] - intervalsHost[aObjId];

	if (nbrCount == 0)
		return;

	const unsigned int nodeCount = nbrCount + 1u;
	thrust::host_vector<unsigned int> nodeIds(nodeCount, aObjId);
	thrust::copy(adjacencyValsHost.begin() + intervalsHost[aObjId], adjacencyValsHost.begin() + intervalsHost[aObjId + 1], nodeIds.begin() + 1u);

	thrust::host_vector<float3> vertexBufferHost;
	thrust::host_vector<uint2> vtxRanges;

	VertexBufferUnpacker unpackVertices;
	unpackVertices(aObj, nodeIds, vtxRanges, vertexBufferHost);



	//Use PCA to compute local coordiante system for each object
	thrust::host_vector<float3> translations(nodeCount);
	thrust::host_vector<quaternion4f> rotations(nodeCount);
	thrust::host_vector<double> tmpCovMatrix(nodeCount * 3 * 3, 0.f);
	thrust::host_vector<double> tmpDiagonalW(nodeCount * 3);
	thrust::host_vector<double> tmpMatrixV(nodeCount * 3 * 3);
	thrust::host_vector<double> tmpVecRV(nodeCount * 3);

	LocalCoordsEstimator estimateT(
		thrust::raw_pointer_cast(vtxRanges.data()),
		thrust::raw_pointer_cast(vertexBufferHost.data()),
		thrust::raw_pointer_cast(tmpCovMatrix.data()),
		thrust::raw_pointer_cast(tmpDiagonalW.data()),
		thrust::raw_pointer_cast(tmpMatrixV.data()),
		thrust::raw_pointer_cast(tmpVecRV.data()),
		thrust::raw_pointer_cast(translations.data()),
		thrust::raw_pointer_cast(rotations.data())
	);

	//thrust::counting_iterator<size_t> first(0u);
	//thrust::counting_iterator<size_t> last(nodeCount);
	//thrust::for_each(first, last, estimateT);

	for (unsigned int i = 0u; i < nodeCount; ++i)
	{
		estimateT(i);
	}

	if (debugOutputLocalFrames)
	{
		transformObj(aObj, nodeIds[0], translations[0], make_float3(0.f, 0.f, 0.f), rotations[0].conjugate());
		return;
	}

	const unsigned int nodeId1 = nodeIds[0];

	for (unsigned int i = 1; i < nodeIds.size(); i++)
	{
		const unsigned int nodeId2 = nodeIds[i];

		if (visited[nodeId2])
			continue;

		const unsigned int typeId1 = nodeTypeIds[nodeId1];
		const unsigned int typeId2 = nodeTypeIds[nodeId2];

		quaternion4f rot = rotations[0];
		float3 relativeT = transformVec(rot.conjugate(), translations[i] - translations[0]);
		quaternion4f relativeR = rotations[i].conjugate() * rot;

		float3 bestT = relativeT;
		quaternion4f bestR = relativeR;
		quaternion4f bestA = relativeR;

		findBestMatch(typeId1, typeId2, relativeT, relativeR, bestT, bestR, bestA);
		const float angleDelta = fabsf(fabsf((bestR * relativeR.conjugate()).w) - 1.f);
		if (angleDelta < angleTolerance)
			continue;
		float3 translateDelta = (0.25f) * transformVec(rot, bestT - relativeT);

		transformObj(aObj, nodeId2, translations[i], translateDelta, rotations[i] * bestR * rot.conjugate());

		++numCorrections;
	}

}

__host__ void Wiggle::findBestMatch(
	unsigned int		aTypeId1,
	unsigned int		aTypeId2,
	const float3&		aTranslation,
	const quaternion4f&	aRotation,
	float3&				oTranslation,
	quaternion4f&		oRotation,
	quaternion4f&		oAbsRotation)
{
	float bestSpatialDist = FLT_MAX;
	for (unsigned int id = mIntervals[aTypeId1]; id < mIntervals[aTypeId1 + 1]; id++)
	{
		if (mNeighborTypeVals[id] != aTypeId2)
			continue;
		const float3 delta = mRelativeTranslation[id] - aTranslation;
		const float currentSpatialDist = len(delta);
		if (currentSpatialDist < bestSpatialDist)
		{
			bestSpatialDist = currentSpatialDist;
		}
	}

	float bestAngleDist = FLT_MAX;
	for (unsigned int id = mIntervals[aTypeId1]; id < mIntervals[aTypeId1 + 1]; id++)
	{
		if (mNeighborTypeVals[id] != aTypeId2)
			continue;
		const float3 delta = mRelativeTranslation[id] - aTranslation;
		const float currentSpatialDist = len(delta);
		const float angleDelta = fabsf(fabsf((aRotation *  mRelativeRotation[id].conjugate()).w) - 1.f);
		if (currentSpatialDist < spatialTolerance + bestSpatialDist && angleDelta < bestAngleDist)
		{
			bestAngleDist = angleDelta;

			oTranslation = mRelativeTranslation[id];
			oRotation = mRelativeRotation[id];
			oAbsRotation = mAbsoluteRotation[id];
		}
	}

}

__host__ void Wiggle::transformObj(
	WFObject & aObj,
	unsigned int aObjId,
	const float3 & aObjCenter,
	const float3 & aTranslation,
	const quaternion4f & aRotation)
{
	thrust::host_vector<unsigned int> processed(aObj.getNumVertices(), 0u);
	for (int faceId = aObj.objects[aObjId].x; faceId < aObj.objects[aObjId].y; ++faceId)
	{
		WFObject::Face face = aObj.faces[faceId];
		size_t vtxId1 = aObj.faces[faceId].vert1;
		size_t vtxId2 = aObj.faces[faceId].vert2;
		size_t vtxId3 = aObj.faces[faceId].vert3;
		if (processed[vtxId1] == 0u)
		{
			processed[vtxId1] = 1u;
			float3 vtx = aObj.vertices[vtxId1];
			aObj.vertices[vtxId1] = transformVec(aRotation, vtx - aObjCenter) + aObjCenter + aTranslation;
		}
		if (processed[vtxId2] == 0u)
		{
			processed[vtxId2] = 1u;
			float3 vtx = aObj.vertices[vtxId2];
			aObj.vertices[vtxId2] = transformVec(aRotation, vtx - aObjCenter) + aObjCenter + aTranslation;

		}
		if (processed[vtxId3] == 0u)
		{
			processed[vtxId3] = 1u;
			float3 vtx = aObj.vertices[vtxId3];
			aObj.vertices[vtxId3] = transformVec(aRotation, vtx - aObjCenter) + aObjCenter + aTranslation;
		}
	}

}