#include "hip/hip_runtime.h"
#include "pch.h"
#include "Wiggle.h"

#include <deque>

#include "WFObjUtils.h"
#include "SVD.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

class LocalCoordsEstimator
{
public:
	uint2*			vertexRanges;
	float3*			vertexBuffer;
	double*			tmpCovMatrix;
	double*			tmpDiagonalW;
	double*			tmpMatrixV;
	double*			tmpVecRV;
	float3*			outTranslation;
	quaternion4f*	outRotation;


	LocalCoordsEstimator(
		uint2*			aRanges,
		float3*			aBuffer,
		double*			aCovMatrix,
		double*			aDiagonalW,
		double*			aMatrixV,
		double*			aVecRV,
		float3*			aOutTranslation,
		quaternion4f*	aOutRot
	) : 
		vertexRanges(aRanges),
		vertexBuffer(aBuffer),
		tmpCovMatrix(aCovMatrix),
		tmpDiagonalW(aDiagonalW),
		tmpMatrixV(aMatrixV),
		tmpVecRV(aVecRV),
		outTranslation(aOutTranslation),
		outRotation(aOutRot)
	{}

	__host__ __device__	void operator()(const size_t& aId)
	{
		const unsigned int objId = (unsigned)aId;


		//Compute the means of the border node locations
		float3 center = make_float3(0.f, 0.f, 0.f);
		uint2 vtxRange = vertexRanges[objId];
		unsigned int vtxCount = vtxRange.y - vtxRange.x;
		float numPoints = (float)vtxCount;

		for (unsigned int vtxId = 0; vtxId < vtxCount; ++vtxId)
		{
			center += vertexBuffer[vtxRange.x + vtxId];
		}
		center /= numPoints;
		
		//Compute covariance matrix
		double* covMat = thrust::raw_pointer_cast(tmpCovMatrix + aId * 3 * 3);
		for (unsigned int vtxId = 0; vtxId < vtxCount; ++vtxId)
		{
			float3 vec1 = vertexBuffer[vtxRange.x + vtxId];

			covMat[0 * 3 + 0] += (double)vec1.x * vec1.x;
			covMat[1 * 3 + 0] += (double)vec1.y * vec1.x;
			covMat[2 * 3 + 0] += (double)vec1.z * vec1.x;

			covMat[0 * 3 + 1] += (double)vec1.x * vec1.y;
			covMat[1 * 3 + 1] += (double)vec1.y * vec1.y;
			covMat[2 * 3 + 1] += (double)vec1.z * vec1.y;

			covMat[0 * 3 + 2] += (double)vec1.x * vec1.z;
			covMat[1 * 3 + 2] += (double)vec1.y * vec1.z;
			covMat[2 * 3 + 2] += (double)vec1.z * vec1.z;
		}

		//Singular Value Decomposition
		double* diag = thrust::raw_pointer_cast(tmpDiagonalW + aId * 3);
		double* vMat = thrust::raw_pointer_cast(tmpMatrixV + aId * 3 * 3);
		double* tmp = thrust::raw_pointer_cast(tmpVecRV + aId * 3);

		svd::svdcmp(covMat, 3, 3, diag, vMat, tmp);

		//Rotation is V * transpose(U)		
		for (unsigned int row = 0; row < 3; ++row)
		{
			for (unsigned int col = 0; col < 3; ++col)
			{
				tmp[col] =
					vMat[row * 3 + 0] * covMat[col * 3 + 0] +
					vMat[row * 3 + 1] * covMat[col * 3 + 1] +
					vMat[row * 3 + 2] * covMat[col * 3 + 2];
			}
			vMat[row * 3 + 0] = tmp[0];
			vMat[row * 3 + 1] = tmp[1];
			vMat[row * 3 + 2] = tmp[2];
		}


		double rotDet = determinantd(
			vMat[0], vMat[3], vMat[6],
			vMat[1], vMat[4], vMat[7],
			vMat[2], vMat[5], vMat[8]
		);

		//if (rotDet < 0.f)
		//{
		//	vMat[0] = -vMat[0];
		//	vMat[1] = -vMat[1];
		//	vMat[2] = -vMat[2];
		//	rotDet = -rotDet;
		//}


		quaternion4f rotation(
			(float)vMat[0], (float)vMat[3], (float)vMat[6],
			(float)vMat[1], (float)vMat[4], (float)vMat[7],
			(float)vMat[2], (float)vMat[5], (float)vMat[8]
		);
		outTranslation[aId] = center;
		outRotation[aId] = rotation;
	}

};


class TransformationExtractor
{
public:
	thrust::device_ptr<unsigned int> nodeTypes;
	
	thrust::device_ptr<unsigned int> outNeighborTypeKeys;
	thrust::device_ptr<unsigned int> outNeighborTypeVals;

	thrust::device_ptr<float3> translation;
	thrust::device_ptr<quaternion4f> rotation;

	thrust::device_ptr<float3> outTranslation;
	thrust::device_ptr<quaternion4f> outRotation;


	TransformationExtractor(
		thrust::device_ptr<unsigned int> aNodeTypes,
		thrust::device_ptr<unsigned int> aOutNbrTypeKeys,
		thrust::device_ptr<unsigned int> aOutNbrTypeVals,
		thrust::device_ptr<float3> aTranslation,
		thrust::device_ptr<quaternion4f> aRotation,
		thrust::device_ptr<float3> aOutTranslation,
		thrust::device_ptr<quaternion4f> aOutRotation
	) :
		nodeTypes(aNodeTypes),
		outNeighborTypeKeys(aOutNbrTypeKeys),
		outNeighborTypeVals(aOutNbrTypeVals),
		translation(aTranslation),
		rotation(aRotation),
		outTranslation(aOutTranslation),
		outRotation(aOutRotation)
	{}

	template <typename Tuple>
	__host__ __device__	void operator()(Tuple t)
	{
		const unsigned int nodeId1 = thrust::get<0>(t);
		const unsigned int nodeId2 = thrust::get<1>(t);
		const unsigned int outId = (unsigned)thrust::get<2>(t);


		outNeighborTypeKeys[outId] = nodeTypes[nodeId1];
		outNeighborTypeVals[outId] = nodeTypes[nodeId2];

		quaternion4f rot = rotation[nodeId1];
		quaternion4f irot = rot.conjugate();
		outTranslation[outId] = transformVec(irot, translation[nodeId2] - translation[nodeId1]);
		outRotation[outId] = rotation[nodeId2] * irot;
	}

};


__host__ void Wiggle::init(WFObject & aObj, Graph & aGraph)
{
	//Unpack and upload the vertex buffer
	thrust::host_vector<uint2> vertexRangesHost;
	thrust::host_vector<float3> vertexBufferHost;

	VertexBufferUnpacker unpackVertices;
	unpackVertices(aObj, vertexRangesHost, vertexBufferHost);

	thrust::device_vector<uint2> vertexRangesDevice(vertexRangesHost);
	thrust::device_vector<float3> vertexBufferDevice(vertexBufferHost);


	//Use PCA to compute local coordiante system for each object
	thrust::device_vector<float3> outTranslation(aObj.getNumObjects());
	thrust::device_vector<quaternion4f> outRotation(aObj.getNumObjects());

	thrust::device_vector<double> tmpCovMatrix(aObj.getNumObjects() * 3 * 3, 0.f);
	thrust::device_vector<double> tmpDiagonalW(aObj.getNumObjects() * 3);
	thrust::device_vector<double> tmpMatrixV(aObj.getNumObjects() * 3 * 3);
	thrust::device_vector<double> tmpVecRV(aObj.getNumObjects() * 3);

	LocalCoordsEstimator estimateT(
		thrust::raw_pointer_cast(vertexRangesDevice.data()),
		thrust::raw_pointer_cast(vertexBufferDevice.data()),
		thrust::raw_pointer_cast(tmpCovMatrix.data()),
		thrust::raw_pointer_cast(tmpDiagonalW.data()),
		thrust::raw_pointer_cast(tmpMatrixV.data()),
		thrust::raw_pointer_cast(tmpVecRV.data()),
		thrust::raw_pointer_cast(outTranslation.data()),
		thrust::raw_pointer_cast(outRotation.data())
	);

	thrust::counting_iterator<size_t> first(0u);
	thrust::counting_iterator<size_t> last(aObj.getNumObjects());

	thrust::for_each(first, last, estimateT);

	//Extract and upload node type information
	thrust::host_vector<unsigned int> nodeTypesHost(aGraph.numNodes(), (unsigned int)aObj.materials.size());
	for (size_t nodeId = 0; nodeId < aObj.objects.size(); ++nodeId)
	{
		size_t faceId = aObj.objects[nodeId].x;
		size_t materialId = aObj.faces[faceId].material;
		nodeTypesHost[nodeId] = (unsigned int)materialId;
	}
	thrust::device_vector<unsigned int> nodeTypes(nodeTypesHost);

	thrust::device_vector<unsigned int> neighborTypeKeys(aGraph.numEdges() * 2u);
	thrust::device_vector<unsigned int> neighborTypeVals(aGraph.numEdges() * 2u);
	thrust::device_vector<float3> relativeTranslation(aGraph.numEdges() * 2u);
	thrust::device_vector<quaternion4f> relativeRotation(aGraph.numEdges() * 2u);

	TransformationExtractor extractRelativeT(
		nodeTypes.data(),
		neighborTypeKeys.data(),
		neighborTypeVals.data(),
		outTranslation.data(),
		outRotation.data(),
		relativeTranslation.data(),
		relativeRotation.data()
	);

	thrust::counting_iterator<size_t> lastEdge(aGraph.numEdges() * 2u);

	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(aGraph.adjacencyKeys.begin(), aGraph.adjacencyVals.begin(), first)),
		thrust::make_zip_iterator(thrust::make_tuple(aGraph.adjacencyKeys.end(), aGraph.adjacencyVals.end(), lastEdge)),
		extractRelativeT);

	if(mNeighborTypeKeys.size() == 0u)
	{ 
		//first call of init
		mNeighborTypeKeys = thrust::host_vector<unsigned int>(neighborTypeKeys);
		mNeighborTypeVals = thrust::host_vector<unsigned int>(neighborTypeVals);
		mRelativeTranslation = thrust::host_vector<float3>(relativeTranslation);
		mRelativeRotation = thrust::host_vector<quaternion4f>(relativeRotation);
	}
	else
	{
		//init already called, append new data
		size_t oldCount = mNeighborTypeKeys.size();
		mNeighborTypeKeys.resize(oldCount + neighborTypeKeys.size());
		mNeighborTypeVals.resize(oldCount + neighborTypeVals.size());
		mRelativeTranslation.resize(oldCount + relativeTranslation.size());
		mRelativeRotation.resize(oldCount + relativeRotation.size());

		thrust::copy(neighborTypeKeys.begin(), neighborTypeKeys.end(), mNeighborTypeKeys.begin() + oldCount);
		thrust::copy(neighborTypeVals.begin(), neighborTypeVals.end(), mNeighborTypeVals.begin() + oldCount);
		thrust::copy(relativeTranslation.begin(), relativeTranslation.end(), mRelativeTranslation.begin() + oldCount);
		thrust::copy(relativeRotation.begin(), relativeRotation.end(), mRelativeRotation.begin() + oldCount);
	}

	//sort by node type
	thrust::sort_by_key(
		mNeighborTypeKeys.begin(),
		mNeighborTypeKeys.end(),
		thrust::make_zip_iterator(thrust::make_tuple(mNeighborTypeVals.begin(), mRelativeTranslation.begin(), mRelativeRotation.begin()))
		);

	//setup search intervals for each node type
	mIntervals.resize(aObj.materials.size() + 1u, 0u);
	for (size_t i = 0u; i < mNeighborTypeKeys.size() - 1u; ++i)
	{
		if (mNeighborTypeKeys[i] < mNeighborTypeKeys[i + 1u])
		{
			mIntervals[mNeighborTypeKeys[i + 1]] = (unsigned)i + 1u;
		}
	}
	//last element
	if (mNeighborTypeKeys.size() > 0u)
		mIntervals[mNeighborTypeKeys[mNeighborTypeKeys.size() - 1u]] = (unsigned)mNeighborTypeKeys.size();

	//fill gaps due to missing node types
	for (size_t i = 1u; i < mIntervals.size(); ++i)
	{
		mIntervals[i] = std::max(mIntervals[i - 1u], mIntervals[i]);
	}
}

__host__ void Wiggle::fixRelativeTransformations(WFObject & aObj, Graph & aGraph)
{
	//Unpack and upload the vertex buffer
	thrust::host_vector<uint2> vertexRangesHost;
	thrust::host_vector<float3> vertexBufferHost;

	VertexBufferUnpacker unpackVertices;
	unpackVertices(aObj, vertexRangesHost, vertexBufferHost);

	thrust::device_vector<uint2> vertexRangesDevice(vertexRangesHost);
	thrust::device_vector<float3> vertexBufferDevice(vertexBufferHost);


	size_t numNodes = aObj.objects.size();
	thrust::host_vector<unsigned int> visited(numNodes, 0u);
	thrust::host_vector<unsigned int> intervalsHost(aGraph.intervals);
	thrust::host_vector<unsigned int> adjacencyValsHost(aGraph.adjacencyVals);

	//Extract and upload node type information
	thrust::host_vector<unsigned int> nodeTypesHost(aGraph.numNodes(), (unsigned int)aObj.materials.size());
	for (size_t nodeId = 0; nodeId < aObj.objects.size(); ++nodeId)
	{
		size_t faceId = aObj.objects[nodeId].x;
		size_t materialId = aObj.faces[faceId].material;
		nodeTypesHost[nodeId] = (unsigned int)materialId;
	}

	std::deque<unsigned int> frontier;
	frontier.push_back(0u);
	visited[0u] = 1u;
	while (!frontier.empty())
	{
		const unsigned int nodeId = frontier.front();
		frontier.pop_front();
		
		processNeighbors(
			aObj,
			nodeId,
			visited,
			intervalsHost,
			adjacencyValsHost,
			nodeTypesHost);

		for (unsigned int nbrId = intervalsHost[nodeId]; nbrId < intervalsHost[nodeId + 1]; ++nbrId)
		{
			const unsigned int nodeId = adjacencyValsHost[nbrId];
			if (visited[nodeId] == 0u)
			{
				frontier.push_back(nodeId);
				visited[nodeId] = 1u;
			}
		}

	}

}

__host__ void Wiggle::processNeighbors(
	WFObject&							aObj,
	unsigned int						aObjId,
	thrust::host_vector<unsigned int>&	visited,
	thrust::host_vector<unsigned int>&	intervalsHost,
	thrust::host_vector<unsigned int>&	adjacencyValsHost,
	thrust::host_vector<unsigned int>&	nodeTypeIds)
{
	const unsigned int nbrCount = intervalsHost[aObjId + 1u] - intervalsHost[aObjId];

	if (nbrCount == 0)
		return;

	const unsigned int nodeCount = nbrCount + 1u;
	thrust::host_vector<unsigned int> nodeIds(nodeCount, aObjId);
	thrust::copy(nodeIds.begin() + 1u, nodeIds.end(), adjacencyValsHost.begin() + intervalsHost[aObjId]);
	
	unsigned int vtxCount = 0u;
	for (unsigned int i = 0; i < nodeIds.size(); i++)
	{
		vtxCount += 3 * (aObj.objects[nodeIds[i]].y - aObj.objects[nodeIds[i]].x);
	}

	//Unpack the vertex buffer
	thrust::host_vector<float3> vertexBufferHost(vtxCount);
	thrust::host_vector<uint2> vtxRanges(nodeCount);
	unsigned int currentVtxId = 0u;
	for (unsigned int i = 0; i < nodeIds.size(); i++)
	{
		vtxRanges[i].x = currentVtxId;
		for (int faceId = aObj.objects[nodeIds[i]].x; faceId < aObj.objects[nodeIds[i]].y; ++faceId)
		{
			WFObject::Face face = aObj.faces[faceId];
			vertexBufferHost[currentVtxId++] = aObj.vertices[aObj.faces[faceId].vert1];
			vertexBufferHost[currentVtxId++] = aObj.vertices[aObj.faces[faceId].vert2];
			vertexBufferHost[currentVtxId++] = aObj.vertices[aObj.faces[faceId].vert3];
		}
		vtxRanges[i].y = currentVtxId;
	}


	//Use PCA to compute local coordiante system for each object
	thrust::host_vector<float3> translations(nodeCount);
	thrust::host_vector<quaternion4f> rotations(nodeCount);
	thrust::host_vector<double> tmpCovMatrix(nodeCount * 3 * 3, 0.f);
	thrust::host_vector<double> tmpDiagonalW(nodeCount * 3);
	thrust::host_vector<double> tmpMatrixV(nodeCount * 3 * 3);
	thrust::host_vector<double> tmpVecRV(nodeCount * 3);

	LocalCoordsEstimator estimateT(
		thrust::raw_pointer_cast(vtxRanges.data()),
		thrust::raw_pointer_cast(vertexBufferHost.data()),
		thrust::raw_pointer_cast(tmpCovMatrix.data()),
		thrust::raw_pointer_cast(tmpDiagonalW.data()),
		thrust::raw_pointer_cast(tmpMatrixV.data()),
		thrust::raw_pointer_cast(tmpVecRV.data()),
		thrust::raw_pointer_cast(translations.data()),
		thrust::raw_pointer_cast(rotations.data())
	);

	thrust::counting_iterator<size_t> first(0u);
	thrust::counting_iterator<size_t> last(nodeCount);

	thrust::for_each(first, last, estimateT);

	for (unsigned int i = 1; i < nodeIds.size(); i++)
	{
		const unsigned int nbrNodeId = nodeIds[i];
		if (visited[nbrNodeId])
			continue;

		const unsigned int nodeId1 = nodeIds[0];
		const unsigned int nodeId2 = nbrNodeId;

		const unsigned int typeId1 = nodeTypeIds[nodeId1];
		const unsigned int typeId2 = nodeTypeIds[nodeId2];

		quaternion4f rot = rotations[nodeId1];
		quaternion4f irot = rot.conjugate();
		float3 relativeT = transformVec(irot, translations[nodeId2] - translations[nodeId1]);
		quaternion4f relativeR = rotations[nodeId2] * irot;

		float3 bestT;
		quaternion4f bestR;

		findBestMatch(typeId1, typeId2, relativeT, relativeR, bestT, bestR);

		if (magnitudeSQR(relativeR * bestR.conjugate()) < angleTolerance)
			continue;

		transformObj(aObj, nodeId2, -translations[i], relativeR.conjugate(), bestR, bestT, irot, translations[0]);
	}

}

__host__ void Wiggle::findBestMatch(
	unsigned int		aTypeId1,
	unsigned int		aTypeId2,
	const float3&		aTranslation,
	const quaternion4f&	aRotation,
	float3&				oTranslation,
	quaternion4f&		oRotation)
{
	float bestSpatialDist = FLT_MAX;
	for (unsigned int id = mIntervals[aTypeId1]; id < mIntervals[aTypeId1 + 1]; id++)
	{
		if (mNeighborTypeVals[id] != aTypeId2)
			continue;
		const float3 delta = mRelativeTranslation[id] - aTranslation;
		const float currentSpatialDist = dot(delta, delta);
		if (currentSpatialDist < bestSpatialDist)
		{
			bestSpatialDist = currentSpatialDist;
			oTranslation = mRelativeTranslation[id];
			oRotation = mRelativeRotation[id];
		}
	}
}

__host__ void Wiggle::transformObj(
	WFObject & aObj,
	unsigned int aObjId,
	const float3 & aTranslation0toB,
	const quaternion4f & aRotationBtoA,
	const quaternion4f & aRotationAtoC,
	const float3 & aTranslationAtoC,
	const quaternion4f & aRotationAto0,
	const float3 & aTranslationAto0)
{
	thrust::host_vector<unsigned int> processed(aObj.getNumVertices(), 0u);
	for (int faceId = aObj.objects[aObjId].x; faceId < aObj.objects[aObjId].y; ++faceId)
	{
		WFObject::Face face = aObj.faces[faceId];
		size_t vtxId1 = aObj.faces[faceId].vert1;
		size_t vtxId2 = aObj.faces[faceId].vert2;
		size_t vtxId3 = aObj.faces[faceId].vert3;
		if (processed[vtxId1] == 0u)
		{
			processed[vtxId1] = 1u;
			float3 vtx = aObj.vertices[vtxId1];
			aObj.vertices[vtxId1] = transformVtx(vtx, aTranslation0toB, aRotationBtoA, aRotationAtoC, aTranslationAtoC, aRotationAto0, aTranslationAto0);
		}
		if (processed[vtxId2] == 0u)
		{
			processed[vtxId2] = 1u;
			float3 vtx = aObj.vertices[vtxId2];
			aObj.vertices[vtxId2] = transformVtx(vtx, aTranslation0toB, aRotationBtoA, aRotationAtoC, aTranslationAtoC, aRotationAto0, aTranslationAto0);

		}
		if (processed[vtxId3] == 0u)
		{
			processed[vtxId3] = 1u;
			float3 vtx = aObj.vertices[vtxId2];
			aObj.vertices[vtxId2] = transformVtx(vtx, aTranslation0toB, aRotationBtoA, aRotationAtoC, aTranslationAtoC, aRotationAto0, aTranslationAto0);
		}
	}

}

__host__ __device__ float3 Wiggle::transformVtx(
	const float3 & aVtx,
	const float3 & aTranslation0toB,
	const quaternion4f & aRotationBtoA,
	const quaternion4f & aRotationAtoC,
	const float3 & aTranslationAtoC,
	const quaternion4f & aRotationAto0,
	const float3 & aTranslationAto0)
{
	const float3 t_b_to_0 = aVtx + aTranslation0toB;
	const float3 r_b_to_a = transformVec(aRotationBtoA, t_b_to_0);
	const float3 r_a_to_c = transformVec(aRotationAtoC, r_b_to_a);
	const float3 t_a_to_c = r_a_to_c + aTranslationAtoC;
	const float3 r_a_to_0 = transformVec(aRotationAto0, t_a_to_c);
	const float3 t_a_to_0 = r_a_to_0 + aTranslationAto0;
	return t_a_to_0;
}
