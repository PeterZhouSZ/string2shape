#include "hip/hip_runtime.h"
#include "pch.h"
#include "WFObjectToString.h"

#include "WFObject.h"
#include "UniformGrid.h"
#include "UniformGridSortBuilder.h"

#include "Graph.h"
#include "CollisionDetector.h"
#include "CollisionGraphExporter.h"
#include "Graph2String.h"
#include "VariationGenerator.h"
#include "Wiggle.h"
#include "PartOrientationUtils.h"
#include "WFObjUtils.h"
#include "RNG.h"
#include "GraphToWFObject.h"

#include "DebugUtils.h"


#ifdef __cplusplus
extern "C" {
#endif
	char * outputString = NULL;

	char * WFObjectToString(const char * aFilename)
	{
		WFObject obj;
		obj.read(aFilename);

		CollisionDetector detector;
		Graph graph = detector.computeCollisionGraph(obj, 0.0f);

		CollisionGraphExporter exporter;
		exporter.exportCollisionGraph(aFilename, obj, graph);


		GraphToStringConverter converter;
		std::string result = converter(obj, graph).first.c_str();
		
		result = result.substr(0u, result.find_first_of("\n"));

		if (outputString != NULL)
			free(outputString);

		outputString = new char[result.length() + 1];
		strcpy(outputString, result.c_str());

		return outputString;
	}

	char * WFObjectToStrings(const char * aFilename, bool aAppendNodeIds/* = false*/)
	{
		WFObject obj;
		obj.read(aFilename);

		CollisionDetector detector;
		Graph graph = detector.computeCollisionGraph(obj, 0.0f);

		CollisionGraphExporter exporter;
		exporter.exportCollisionGraph(aFilename, obj, graph);

		if (!graph.isConnected())
			return "";

		GraphToStringConverter converter;
		std::pair< std::string, std::string > strings_nodeIds = converter(obj, graph);

		for (size_t i = 0; i < 10; ++i)
		{
			std::pair< std::string, std::string > new_sample = converter(obj, graph);
			strings_nodeIds.first.append(new_sample.first);
			strings_nodeIds.second.append(new_sample.second);
		}

		std::string result = strings_nodeIds.first;

		if (aAppendNodeIds)
			result.append(strings_nodeIds.second.substr(0u, strings_nodeIds.second.find_last_of("\n")));

		if (outputString != NULL)
			free(outputString);

		outputString = new char[result.length() + 1];
		strcpy(outputString, result.c_str());

		return outputString;
	}

	char * WFObjectRandomVariations(const char * aFileName1, const char* aFileName2)
	{
		WFObject obj1;
		obj1.read(aFileName1);

		WFObject obj2;
		obj2.read(aFileName2);

		CollisionDetector detector;
		Graph graph1 = detector.computeCollisionGraph(obj1, 0.0f);
		Graph graph2 = detector.computeCollisionGraph(obj2, 0.0f);

		VariationGenerator genRandVariation;
		genRandVariation.writeVariationGraphs = false;
		genRandVariation.writeVariations = true;
		//genRandVariation.fixVariation = true;

		std::string result = genRandVariation(aFileName1, aFileName2, obj1, obj2, graph1, graph2, 0.0f);

		outputString = new char[result.length() + 1];
		strcpy(outputString, result.c_str());

		return outputString;
	}

	int buildGrid(const char * aFilename, int aResX, int aResY, int aResZ)
	{
		WFObject testObj;
		testObj.read(aFilename);

		UniformGridSortBuilder builder;
		UniformGrid grid = builder.build(testObj, aResX, aResY, aResZ);
		builder.stats();

		int result = builder.test(grid, testObj);
		grid.cleanup();

		return result;
	}

	int testGraphConstruction(int aGraphSize)
	{
		Graph testGraph;
		return testGraph.testGraphConstruction(aGraphSize);
	}

	int testCollisionGraphConstruction(const char * aFilename)
	{
		WFObject testObj;
		testObj.read(aFilename);

		CollisionDetector detector;
		Graph testGraph = detector.computeCollisionGraph(testObj, 0.02f);
		detector.stats();		

		CollisionGraphExporter exporter;
		exporter.exportCollisionGraph(aFilename, testObj, testGraph);
		exporter.stats();

		return testGraph.testSpanningTreeConstruction();
	}

	int testRandomVariations(const char * aFileName1, const char* aFileName2)
	{
		WFObject obj1;
		obj1.read(aFileName1);

		WFObject obj2;
		obj2.read(aFileName2);

		CollisionDetector detector;
		Graph graph1 = detector.computeCollisionGraph(obj1, 0.0f);
		Graph graph2 = detector.computeCollisionGraph(obj2, 0.0f);

		CollisionGraphExporter exporter;
		exporter.exportCollisionGraph(aFileName1, obj1, graph1);
		exporter.stats();

		exporter.exportCollisionGraph(aFileName2, obj2, graph2);
		exporter.stats();

		VariationGenerator genRandVariation;
		genRandVariation.writeVariations = true;
		//genRandVariation.fixVariation = true;
		genRandVariation(aFileName1, aFileName2, obj1, obj2, graph1, graph2, 0.0f);
		genRandVariation.stats();

		return 0;

	}

	int fixVariation(const char * aFileName1, const char* aFileName2, const char* aFileName3, const char* aOutFileName)
	{
		WFObject obj1;
		obj1.read(aFileName1);

		WFObject obj2;
		obj2.read(aFileName2);

		WFObject obj3;
		obj3.read(aFileName3);

		CollisionDetector detector;
		Graph graph1 = detector.computeCollisionGraph(obj1, 0.0f);
		Graph graph2 = detector.computeCollisionGraph(obj2, 0.0f);
		Graph graph3 = detector.computeCollisionGraph(obj3, 0.0f);

		GrammarCheck grammarCheck;
		grammarCheck.init(obj1, graph1.intervals, graph1.adjacencyVals);
		grammarCheck.init(obj2, graph2.intervals, graph2.adjacencyVals);

		thrust::host_vector<unsigned int> nodeTypes(graph3.numNodes());
		for (size_t nodeId = 0; nodeId < graph3.numNodes(); ++nodeId)
		{
			size_t faceId = obj3.objects[nodeId].x;
			size_t materialId = obj3.faces[faceId].material;
			nodeTypes[nodeId] = (unsigned int)materialId;
		}
		thrust::host_vector<unsigned int> hostIntervals(graph3.intervals);
		thrust::host_vector<unsigned int> hostNbrIds(graph3.adjacencyVals);
		if (!grammarCheck.check(hostIntervals, hostNbrIds, nodeTypes))
		{
			//std::cerr << "Invalid repair target - does not conform grammar.\n";
			return 1;
		}


		Wiggle wiggle;
		wiggle.init(obj1, graph1);
		wiggle.init(obj2, graph2);
		//wiggle.debugOutputLocalFrames = true;

		for (size_t i = 0; i < 128u; ++i)
		{
			wiggle.fixRelativeTransformations(obj3, graph3);
			if (wiggle.numCorrections == 0u)
				break;
		}

		Graph modifiedGraph = detector.computeCollisionGraph(obj3, 0.0f);
		hostIntervals = thrust::host_vector<unsigned int>(modifiedGraph.intervals);
		hostNbrIds = thrust::host_vector<unsigned int>(modifiedGraph.adjacencyVals);

		if (wiggle.debugOutputLocalFrames || grammarCheck.check(hostIntervals, hostNbrIds, nodeTypes))
		{
			std::string outFileName(aOutFileName);
			if (outFileName.find_last_of("/\\") == std::string::npos)
				outFileName = outFileName.substr(0, outFileName.size() - 5);
			else
				outFileName = outFileName.substr(outFileName.find_last_of("/\\") + 1, outFileName.size() - outFileName.find_last_of("/\\") - 5);

			std::string objDir = getDirName(aFileName3);
			std::string fixedFilePath = objDir + outFileName;

			WFObjectFileExporter   objExporter;
			objExporter(obj3, fixedFilePath.c_str());
		}
		else
		{
			//std::cerr << "Object repair attempt failed.\n";
			return 2;
		}

		return 0;

	}

	int StringToWFObject(
		const char * aFileName1,
		const char * aFileName2,
		const char * aInputString,
		const char * aOutFileName)
	{
		WFObject obj1;
		obj1.read(aFileName1);

		WFObject obj2;
		obj2.read(aFileName2);

		CollisionDetector detector;
		Graph graph1 = detector.computeCollisionGraph(obj1, 0.0f);
		Graph graph2 = detector.computeCollisionGraph(obj2, 0.0f);

		std::stringstream ss(aInputString);
		std::string NodesStr_1_1;
		std::getline(ss, NodesStr_1_1, '\n');
		std::string NodesStr_1_2;
		std::getline(ss, NodesStr_1_2, '\n');
		std::string EdgeTypeStr_1;
		std::getline(ss, EdgeTypeStr_1, '\n');

		thrust::host_vector<unsigned int> keys1(graph1.adjacencyKeys);
		thrust::host_vector<unsigned int> vals1(graph1.adjacencyVals);
		thrust::host_vector<unsigned int> edgeTypes1(graph1.numEdges() * 2, (unsigned int)-1);

		std::stringstream nodes_ss_1_1(NodesStr_1_1);
		std::stringstream nodes_ss_1_2(NodesStr_1_2);
		std::stringstream edge_types_1(EdgeTypeStr_1);
		unsigned int node_1, node_2, edge_type;
		while (nodes_ss_1_1 >> node_1 && nodes_ss_1_2 >> node_2 && edge_types_1 >> edge_type)
		{
			for (unsigned int edgeId = 0u; edgeId < graph1.numEdges() * 2; ++edgeId)
			{
				if (keys1[edgeId] == node_1 && vals1[edgeId] == node_2)
				{
					edgeTypes1[edgeId] = edge_type;
					break;
				}
			}
		}

		//outputHostVector("keys1     : ", keys1);
		//outputHostVector("vals1     : ", vals1);
		//outputHostVector("edgeTypes1: ", edgeTypes1);

		std::string NodesStr_2_1;
		std::getline(ss, NodesStr_2_1, '\n');
		std::string NodesStr_2_2;
		std::getline(ss, NodesStr_2_2, '\n');
		std::string EdgeTypeStr_2;
		std::getline(ss, EdgeTypeStr_2, '\n');

		thrust::host_vector<unsigned int> keys2(graph2.adjacencyKeys);
		thrust::host_vector<unsigned int> vals2(graph2.adjacencyVals);
		thrust::host_vector<unsigned int> edgeTypes2(graph2.numEdges() * 2, (unsigned int)-1);
		std::stringstream nodes_ss_2_1(NodesStr_2_1);
		std::stringstream nodes_ss_2_2(NodesStr_2_2);
		std::stringstream edge_types_2(EdgeTypeStr_2);
		while (nodes_ss_2_1 >> node_1 && nodes_ss_2_2 >> node_2 && edge_types_2 >> edge_type)
		{
			for (unsigned int edgeId = 0u; edgeId < graph2.numEdges() * 2; ++edgeId)
			{
				if (keys2[edgeId] == node_1 && vals2[edgeId] == node_2)
				{
					edgeTypes2[edgeId] = edge_type;
					break;
				}
			}
		}

		//outputHostVector("keys2     : ", keys2);
		//outputHostVector("vals2     : ", vals2);
		//outputHostVector("edgeTypes2: ", edgeTypes2);

		std::string NodesStr_3_1;
		std::getline(ss, NodesStr_3_1, '\n');
		std::string NodesStr_3_2;
		std::getline(ss, NodesStr_3_2, '\n');
		std::string EdgeTypeStr_3;
		std::getline(ss, EdgeTypeStr_3, '\n');

		std::vector<unsigned int> keys3;
		std::vector<unsigned int> vals3;
		std::vector<unsigned int> edgeTypes3;
		std::stringstream nodes_ss_3_1(NodesStr_3_1);
		std::stringstream nodes_ss_3_2(NodesStr_3_2);
		std::stringstream edge_types_3(EdgeTypeStr_3);
		unsigned int max_node_id = 0u;
		while (nodes_ss_3_1 >> node_1 && nodes_ss_3_2 >> node_2 && edge_types_3 >> edge_type)
		{
			max_node_id = std::max(max_node_id, node_1);
			max_node_id = std::max(max_node_id, node_2);
			keys3.push_back(node_1);
			vals3.push_back(node_2);
			edgeTypes3.push_back(edge_type);
		}
		
		Graph graph3;
		graph3.adjacencyKeys = thrust::device_vector<unsigned int>(keys3);
		graph3.adjacencyVals = thrust::device_vector<unsigned int>(vals3);
		graph3.fromAdjacencyList(max_node_id + 1, false);

		thrust::host_vector<unsigned int> edgeTypes3Host(graph3.numEdges() * 2, (unsigned)-1);
		for (unsigned int i = 0u; i < edgeTypes3.size(); ++i)
		{
			unsigned int nodeId_1 = keys3[i];
			unsigned int nodeId_2 = vals3[i];
			unsigned int edgeTypeId = edgeTypes3[i];
			unsigned int edgeId = graph3.getEdgeId(nodeId_1, nodeId_2);
			edgeTypes3Host[edgeId] = edgeTypeId;
		}

		//outputDeviceVector("keys3     : ", graph3.adjacencyKeys);
		//outputDeviceVector("vals3     : ", graph3.adjacencyVals);
		//outputHostVector("edgeTypes3: ", edgeTypes3Host);

		WFObjectGenerator embedGraphAsObj;

		///////////////////////////
		//check edge configurations
		///////////////////////////
		bool allEdgeTypesFound = true;
		for (unsigned int edgeId3 = 0; edgeId3 < edgeTypes3Host.size(); ++edgeId3)
		{
			unsigned int edgeTypeId3 = edgeTypes3Host[edgeId3];
			unsigned int node3A = graph3.adjacencyKeys[edgeId3];
			unsigned int node3B = graph3.adjacencyVals[edgeId3];

			unsigned int oposingEdgeId3 = graph3.getOpositeEdgeId(edgeId3);
			unsigned int oposingTypeId3 = edgeTypes3Host[oposingEdgeId3];
			
			bool foundIn1 = false;
			for (unsigned int edgeId1 = 0; edgeId1 < edgeTypes1.size(); ++edgeId1)
			{
				unsigned int edgeTypeId1 = edgeTypes1[edgeId1];

				unsigned int oposingEdgeId1 = graph1.getOpositeEdgeId(edgeId1);
				unsigned int oposingTypeId1 = edgeTypes1[oposingEdgeId1];

				if (edgeTypeId3 == edgeTypeId1 && oposingTypeId3 == oposingTypeId1)
				{
					foundIn1 = true;
					break;
				}
			}
			bool foundIn2 = false;
			for (unsigned int edgeId2 = 0; edgeId2 < edgeTypes2.size(); ++edgeId2)
			{
				unsigned int edgeTypeId2 = edgeTypes2[edgeId2];

				unsigned int oposingEdgeId2 = graph2.getOpositeEdgeId(edgeId2);
				unsigned int oposingTypeId2 = edgeTypes2[oposingEdgeId2];

				if (edgeTypeId3 == edgeTypeId2 && oposingTypeId3 == oposingTypeId2)
				{
					foundIn2 = true;
					break;
				}
			}
			if (!foundIn1 && !foundIn2)
			{
				allEdgeTypesFound = false;
				//std::cerr << "Did not find type pair " << edgeTypeId3 << " and " << oposingTypeId3 << "\n";
				//std::cerr << "Node ids in target graph " << node3A << " and " << node3B << "\n";							
			}
		}

		if(!allEdgeTypesFound)
			std::cerr << "Did not find examples for some of the requested edge type pairs in the input shapes.\n";

		GrammarCheck grammarCheck;
		grammarCheck.init(obj1, graph1.intervals, graph1.adjacencyVals);
		grammarCheck.init(obj2, graph2.intervals, graph2.adjacencyVals);

		for (size_t attempt = 0; attempt < 16; ++attempt)
		{
			WFObject obj3 = embedGraphAsObj(obj1, obj2, graph1, graph2, graph3, edgeTypes1, edgeTypes2, edgeTypes3Host);
			
			if (obj3.getNumObjects() <= 0)
				continue;

			Graph graph3_1 = detector.computeCollisionGraph(obj3, 0.0f);
			thrust::host_vector<unsigned int> nodeTypes(graph3_1.numNodes());
			for (size_t nodeId = 0; nodeId < graph3_1.numNodes(); ++nodeId)
			{
				size_t faceId = obj3.objects[nodeId].x;
				size_t materialId = obj3.faces[faceId].material;
				nodeTypes[nodeId] = (unsigned int)materialId;
			}
			thrust::host_vector<unsigned int> hostIntervals(graph3_1.intervals);
			thrust::host_vector<unsigned int> hostNbrIds(graph3_1.adjacencyVals);

			if (grammarCheck.check(hostIntervals, hostNbrIds, nodeTypes))
			{
				WFObjectFileExporter()(obj3, aOutFileName);
				if(obj3.getNumObjects() == graph3.numNodes())
					return 0;
			}
		}

		embedGraphAsObj.strictEmbeddingFlag = false;
		for (size_t attempt = 0; attempt < 16; ++attempt)
		{
			WFObject obj3 = embedGraphAsObj(obj1, obj2, graph1, graph2, graph3, edgeTypes1, edgeTypes2, edgeTypes3Host);
			
			if (obj3.getNumObjects() <= 0)
				continue;

			Graph graph3_1= detector.computeCollisionGraph(obj3, 0.0f);
			thrust::host_vector<unsigned int> nodeTypes(graph3_1.numNodes());
			for (size_t nodeId = 0; nodeId < graph3_1.numNodes(); ++nodeId)
			{
				size_t faceId = obj3.objects[nodeId].x;
				size_t materialId = obj3.faces[faceId].material;
				nodeTypes[nodeId] = (unsigned int)materialId;
			}
			thrust::host_vector<unsigned int> hostIntervals(graph3_1.intervals);
			thrust::host_vector<unsigned int> hostNbrIds(graph3_1.adjacencyVals);

			if (grammarCheck.check(hostIntervals, hostNbrIds, nodeTypes))
			{
				std::cerr << "Found a valid graph embedding using not strictly matching edge category pairs.\n";
				WFObjectFileExporter()(obj3, aOutFileName);
				return 0;
			}
		}

		std::string lastAttemptObjFileName = std::string(aOutFileName) + std::string("_attempt");
		
		//embedGraphAsObj.strictEmbeddingFlag = true;
		WFObject obj3 = embedGraphAsObj(obj1, obj2, graph1, graph2, graph3, edgeTypes1, edgeTypes2, edgeTypes3Host);

		//Wiggle wiggle;
		//wiggle.init(obj1, graph1);
		//wiggle.init(obj2, graph2);
		////wiggle.debugOutputLocalFrames = true;

		//for (size_t i = 0; i < 128u; ++i)
		//{
		//	wiggle.fixRelativeTransformations(obj3, graph3);
		//	if (wiggle.numCorrections == 0u)
		//		break;
		//}

		WFObjectFileExporter()(obj3, lastAttemptObjFileName.c_str());

		//std::cerr << "Failed to find a valid shape embedding. Writing an attempt in " << lastAttemptObjFileName << ".obj\n";

		return 1;
	}


	int testRandomNumberGenerator()
	{
		const unsigned int aConst1 = (unsigned int)std::chrono::system_clock::now().time_since_epoch().count();
		const unsigned int aConst2 = 22;
		for (unsigned int tId = 0u; tId < 500u; ++tId)
		{
			//KISSRandomNumberGenerator genRand(
			//	3643u + aConst1 + aConst2 * aConst2,
			//	aConst1,
			//	331801u + aConst2 * aConst1,
			//	10499029u);

			XorShift32Plus genRand(aConst1 + aConst2, aConst2 * aConst2 * aConst1);

			bool between0000_0125 = false;
			bool between0125_0025 = false;
			bool between0250_0375 = false;
			bool between0375_0500 = false;
			bool between0500_0625 = false;
			bool between0625_0750 = false;
			bool between0750_0875 = false;
			bool between0875_1000 = false;
			for (unsigned int testId = 0u; testId < 10000u; ++testId)
			{
				const float r = genRand();
				if (r < -EPS)
				{
					std::cerr << "Random number " << r << " < 0\n";
					return 1;
				}

				if (r > 1.f + EPS)
				{
					std::cerr << "Random number " << r << " > 1\n";
					return 2;
				}

				if (r < 0.125f)
				{
					between0000_0125 = true;
				}
				else if (r < 0.25f)
				{
					between0125_0025 = true;
				}
				else if (r < 0.375f)
				{
					between0250_0375 = true;
				}
				else if (r < 0.5f)
				{
					between0375_0500 = true;
				}
				else if (r < 0.625f)
				{
					between0500_0625 = true;
				}
				else if (r < 0.75f)
				{
					between0625_0750 = true;
				}
				else if (r < 0.875f)
				{
					between0750_0875 = true;
				}
				else
				{
					between0875_1000 = true;
				}
			}

			if (!(between0000_0125
				&& between0125_0025
				&& between0250_0375
				&& between0375_0500
				&& between0500_0625
				&& between0625_0750
				&& between0750_0875
				&& between0875_1000))
			{
				std::cerr << "Failed to cover each of eight bins [0,1] with 10000 samples.\n";
				return 3;
			}
		}


		return 0;

	}
#ifdef __cplusplus
}
#endif

std::vector<float> WFObjectToGraph(const char * aFilename)
{
	WFObject obj;
	obj.read(aFilename);

	CollisionDetector detector;
	Graph graph = detector.computeCollisionGraph(obj, 0.0f);

	PartOrientationEstimator orientationEstimator;
	orientationEstimator.init(obj, graph);

	return orientationEstimator.getEdgesTypesAndOrientations();
}
