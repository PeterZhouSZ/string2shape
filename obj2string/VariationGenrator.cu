#include "hip/hip_runtime.h"
#include "pch.h"
#include "VariationGenerator.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/scan.h>

#include "Algebra.h"
#include "SVD.h"
#include "WFObjUtils.h"
#include "Graph2String.h"
#include "CollisionDetector.h"
#include "CollisionGraphExporter.h"

#include "DebugUtils.h"
#include "Timer.h"


class DistanceMatrixWriter
{
public:
	size_t stride;
	thrust::device_ptr<float> matrix;
	thrust::device_ptr<float3> positions;


	DistanceMatrixWriter(
		size_t aStride,
		thrust::device_ptr<float> aMatrix,
		thrust::device_ptr<float3> aPositions
	) :stride(aStride), matrix(aMatrix), positions(aPositions)
	{}

	__host__ __device__	void operator()(const size_t& aId)
	{
		const size_t myRowId = aId % stride;
		const size_t myColId = aId / stride;
		const float3 objCenter1 = positions[myColId];
		const float3 objCenter2 = positions[myRowId];
		matrix[myColId + myRowId * stride] = len(objCenter1 - objCenter2);
	}

};

class SubgraphInitializer
{
public:
	unsigned int graphSize;
	unsigned int subgraphSize;
	unsigned int numSubgraphs;
	unsigned int subgraphsPerSeedNode;
	//graph edges
	thrust::device_ptr<unsigned int> adjIntervals;
	thrust::device_ptr<unsigned int> neighborIds;

	thrust::device_ptr<unsigned int> outNodeIds;
	thrust::device_ptr<unsigned int> outBorderNodeFlags;

	SubgraphInitializer(
		unsigned int aGraphSize,
		unsigned int aSampleSize,
		unsigned int aNumSamples,
		thrust::device_ptr<unsigned int> aIntervals,
		thrust::device_ptr<unsigned int> aNeighborIds,
		thrust::device_ptr<unsigned int> outIds,
		thrust::device_ptr<unsigned int> outFlags
		) : graphSize(aGraphSize),
		subgraphSize(aSampleSize),
		numSubgraphs(aNumSamples),
		subgraphsPerSeedNode(aNumSamples / aGraphSize),
		adjIntervals(aIntervals),
		neighborIds(aNeighborIds),
		outNodeIds(outIds),
		outBorderNodeFlags(outFlags)
	{}

	__host__ __device__	void operator()(const size_t& aId_s)
	{
		unsigned int aId = (unsigned int)aId_s;

		KISSRandomNumberGenerator genRand(
			3643u + aId * 4154207u * subgraphsPerSeedNode + numSubgraphs,
			1761919u + aId * 2746753u * subgraphsPerSeedNode ,
			331801u + aId,
			10499029u);

		unsigned int subgraphSeedNodeId = subgraphsPerSeedNode == 0u ? aId : aId / subgraphsPerSeedNode;
		//unsigned int subgraphSeedNodeId = max(min((unsigned int)(genRand() * (float)graphSize), graphSize - 1u), 0u);
		unsigned int subgraphStartLocation = aId * subgraphSize;

		outNodeIds[subgraphStartLocation] = subgraphSeedNodeId;

		unsigned int currentSize = 1u;
		//unsigned int globalNbrCount = 0u;

		//compute subgraph
		for(unsigned int currentDepth = 0u; currentDepth < subgraphSize && currentSize < subgraphSize && currentSize < graphSize; ++currentDepth)
		{
			unsigned int neighborCount = 0u;
			for (unsigned int localNodeId = 0u; localNodeId < currentSize; ++localNodeId)
			{
				unsigned int nodeId = outNodeIds[subgraphStartLocation + localNodeId];
				//const float numNbrsRCP = 1.f / (float)(adjIntervals[nodeId + 1u] - adjIntervals[nodeId]);
				for (unsigned int localNeighborId = adjIntervals[nodeId]; localNeighborId < adjIntervals[nodeId + 1u]; ++localNeighborId)
				{
					unsigned int neighborId = neighborIds[localNeighborId];					
					bool alreadyIncluded = false;
					for (unsigned int previousNodeId = 0u; previousNodeId < currentSize + neighborCount; ++previousNodeId)
					{						
						if (outNodeIds[subgraphStartLocation + previousNodeId] == neighborId)
						{
							alreadyIncluded = true;
							break;
						}
					}

					//if (!alreadyIncluded && subgraphsPerSeedNode > 0u)
					//{
					//	//deterministically discard the neighbor
					//	unsigned int subgraphSampleId = aId % subgraphsPerSeedNode;
					//	unsigned int includeFlag = (subgraphSampleId >> globalNbrCount) & 0x00000001;
					//	if (includeFlag != 0u)
					//	{							
					//		alreadyIncluded = true; 
					//	}
					//	++globalNbrCount;
					//}

					if (!alreadyIncluded)
					{
						//randomly discard the neighbor node
						alreadyIncluded = genRand() < 0.5f; // numNbrsRCP;
					}

					if (!alreadyIncluded && neighborCount + currentSize < subgraphSize) //add to subgraph
					{
						outNodeIds[subgraphStartLocation + neighborCount + currentSize] = neighborId;
						neighborCount++;
					}
					else if (!alreadyIncluded && neighborCount > 0u  && genRand() < 0.5)//replace a random node with the same depth
					{
						unsigned int randLocation = (int)(genRand() * (float)(neighborCount));
						outNodeIds[subgraphStartLocation + randLocation + currentSize] = neighborId;
					}
				}

			}

			currentSize += neighborCount;
		}

		//compute subgraph interior and border
		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
		{
			unsigned int nodeId = outNodeIds[subgraphStartLocation + localNodeId];
			bool allNeighborsIncluded = true;
			for (unsigned int localNeighborId = adjIntervals[nodeId]; localNeighborId < adjIntervals[nodeId + 1u]; ++localNeighborId)
			{
				unsigned int neighborId = neighborIds[localNeighborId];
				bool alreadyIncluded = false;
				for (unsigned int previousNodeId = 0u; previousNodeId < subgraphSize; ++previousNodeId)
				{
					if (outNodeIds[subgraphStartLocation + previousNodeId] == neighborId)
					{
						alreadyIncluded = true;
						break;
					}
				}
				if (!alreadyIncluded)
				{
					allNeighborsIncluded = false;
					break;
				}
			}

			if (allNeighborsIncluded)
			{
				outBorderNodeFlags[subgraphStartLocation + localNodeId] = 0u;//mark as interior
			}
			else
			{
				outBorderNodeFlags[subgraphStartLocation + localNodeId] = 1u;//mark as border node
			}
		}

		//compute inner border
		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
		{
			unsigned int nodeId = outNodeIds[subgraphStartLocation + localNodeId];
			bool nextToBorder = false;
			for (unsigned int localNeighborId = adjIntervals[nodeId]; localNeighborId < adjIntervals[nodeId + 1u] && !nextToBorder; ++localNeighborId)
			{
				unsigned int neighborId = neighborIds[localNeighborId];
				for (unsigned int previousNodeId = 0u; previousNodeId < subgraphSize; ++previousNodeId)
				{
					if (outNodeIds[subgraphStartLocation + previousNodeId] == neighborId &&
						outBorderNodeFlags[subgraphStartLocation + previousNodeId] == 1u)
					{
						outBorderNodeFlags[subgraphStartLocation + localNodeId] = 2u;//mark as inner border node
						nextToBorder = true;
						break;
					}
				}
			}
		}//end for each node in the subgraph

	}//end operator()

};

class CutMatching
{
public:
	unsigned int graphSize1;
	unsigned int graphSize2;
	unsigned int subgraphSize;
	unsigned int numSubgraphs;
	unsigned int subgraphsPerSeedNode;
	float spatialTolerance;
	//graph 1
	thrust::device_ptr<unsigned int> inNodeTypes;
	//subrgaphs 1
	thrust::device_ptr<unsigned int> inNodeIds;
	thrust::device_ptr<unsigned int> inBorderNodeFlags;
	//node-node distances 1
	thrust::device_ptr<float>        inDistMatrix;
	//node sizes 1
	thrust::device_ptr<float>        inNodeSizes;
	//graph 2
	thrust::device_ptr<unsigned int> outNodeTypes;
	//subrgaphs 2
	thrust::device_ptr<unsigned int> outNodeIds;
	thrust::device_ptr<unsigned int> outBorderNodeFlags;
	//node-node distances 2
	thrust::device_ptr<float>        outDistMatrix;
	//node types 2
	thrust::device_ptr<float>        outNodeSizes;

	thrust::device_ptr<unsigned int> outValidSubgraphFlags;

	CutMatching(
		unsigned int aGraphSize1,
		unsigned int aGraphSize2,
		unsigned int aSampleSize,
		unsigned int aNumSamples,
		float aSpatialTolerance,
		thrust::device_ptr<unsigned int> inTypes,
		thrust::device_ptr<unsigned int> inIds,
		thrust::device_ptr<unsigned int> inFlags,
		thrust::device_ptr<float> inMatrix,
		thrust::device_ptr<float> inSizes,
		thrust::device_ptr<unsigned int> outTypes,
		thrust::device_ptr<unsigned int> outIds,
		thrust::device_ptr<unsigned int> outFlags,
		thrust::device_ptr<float> outMatrix,
		thrust::device_ptr<float> outSizes,
		thrust::device_ptr<unsigned int> outSubgraphFlags
	) : graphSize1(aGraphSize1),
		graphSize2(aGraphSize2),
		subgraphSize(aSampleSize),
		numSubgraphs(aNumSamples),
		subgraphsPerSeedNode(aNumSamples / aGraphSize1),
		spatialTolerance(aSpatialTolerance),
		inNodeTypes(inTypes),
		inNodeIds(inIds),
		inBorderNodeFlags(inFlags),
		inDistMatrix(inMatrix),
		inNodeSizes(inSizes),
		outNodeTypes(outTypes),
		outNodeIds(outIds),
		outBorderNodeFlags(outFlags),
		outDistMatrix(outMatrix),
		outNodeSizes(outSizes),
		outValidSubgraphFlags(outSubgraphFlags)
	{}
	
	//__host__ __device__ FORCE_INLINE void invalidateSubgraph(unsigned int subgraphStartLocation)
	//{
	//	for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
	//	{
	//		outNodeIds[subgraphStartLocation + localNodeId] = graphSize2;
	//		outBorderNodeFlags[subgraphStartLocation + localNodeId] = 0u;
	//	}
	//}

	__host__ __device__	bool randomMatchingOperator(unsigned int aId, unsigned int aRandOffset)
	{
		unsigned int subgraphStartLocation = aId * subgraphSize;// (aId % 32) * subgraphSize;

		unsigned int interiorNodesCount = 0;
		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
		{
			if (inBorderNodeFlags[subgraphStartLocation + localNodeId] == 0u)
			{
				//only match the nodes on the inner border and border
				outNodeIds[subgraphStartLocation + localNodeId] = graphSize2;
				++interiorNodesCount;
			}
			else
			{
				outNodeIds[subgraphStartLocation + localNodeId] = aRandOffset;
			}
			outBorderNodeFlags[subgraphStartLocation + localNodeId] = 0u;
		}

		if (interiorNodesCount == 0 || subgraphSize - interiorNodesCount < 3)
		{
			//invalid subgraph - too few nodes in the cut, or no interior nodes
			//invalidateSubgraph(subgraphStartLocation);
			return false;
		}


		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
		{
			if (inBorderNodeFlags[subgraphStartLocation + localNodeId] == 0u)
				continue;

			unsigned int inNodeId = inNodeIds[subgraphStartLocation + localNodeId];
			unsigned int inNodeType = inNodeTypes[inNodeId];
			bool foundAMatch = false;
			for (unsigned int nodeId2 = outNodeIds[subgraphStartLocation + localNodeId]; nodeId2 < graphSize2 && !foundAMatch; ++nodeId2)
			{
				//unsigned int nodeId2RND = (randNodeIdOffset + nodeId2) % graphSize2;
				//unsigned int nodeId2 = (nodeId2_it + 16) % graphSize2;
				bool matches = outNodeTypes[nodeId2] == inNodeType;
				if (!matches)
					continue;
				bool skip = false;
				for (unsigned int recordedNodeId = 0; recordedNodeId < localNodeId && !skip; ++recordedNodeId)
				{
					if (outNodeIds[subgraphStartLocation + recordedNodeId] == nodeId2)
						skip = true;//already participates
					if (inBorderNodeFlags[subgraphStartLocation + recordedNodeId] == 0u)
						continue;//skip interior nodes
					unsigned int pairNodeId = inNodeIds[subgraphStartLocation + recordedNodeId];
					const float targetDist = inDistMatrix[inNodeId + graphSize1 * pairNodeId];
					unsigned int pairNodeId2 = outNodeIds[subgraphStartLocation + recordedNodeId];
					const float currentDist = outDistMatrix[nodeId2 + graphSize2 * pairNodeId2];
					if (fabsf(currentDist - targetDist) > spatialTolerance /** outNodeSizes[pairNodeId2]*/)
						skip = true;//incompatible with previous participants
				}
				if (skip)
					continue;

				for (unsigned int pairingNodeId = 0; pairingNodeId < subgraphSize; ++pairingNodeId)
				{
					if (inBorderNodeFlags[subgraphStartLocation + pairingNodeId] == 0u)
						continue;
					if (pairingNodeId == localNodeId)
						continue;
					unsigned int pairNodeId = inNodeIds[subgraphStartLocation + pairingNodeId];
					unsigned int pairNodeType = inNodeTypes[pairNodeId];
					const float targetDist = inDistMatrix[inNodeId + graphSize1 * pairNodeId];
					bool foundMatchingPair = false;
					for (unsigned int pairingNodeId2 = 0u; pairingNodeId2 < graphSize2 && !foundMatchingPair; ++pairingNodeId2)
					{
						if (pairingNodeId2 == nodeId2)
							continue;
						if (outNodeTypes[pairingNodeId2] != pairNodeType)
							continue;
						const float currentDist = outDistMatrix[nodeId2 + graphSize2 * pairingNodeId2];
						if (fabsf(currentDist - targetDist) < spatialTolerance /** inNodeSizes[pairingNodeId2]*/)
							foundMatchingPair = true;
					}//end for all other nodes in the second graph
					if (!foundMatchingPair)
						matches = false;
				}//end for all other nodes in the cut
				if (matches)
				{
					outNodeIds[subgraphStartLocation + localNodeId] = nodeId2;
					outBorderNodeFlags[subgraphStartLocation + localNodeId] = inBorderNodeFlags[subgraphStartLocation + localNodeId];
					foundAMatch = true;
				}
			}//end for all nodes in the second graph
			if (!foundAMatch)
			{
				//backtrack
				if (localNodeId > 0) --localNodeId;
				while (inBorderNodeFlags[subgraphStartLocation + localNodeId] == 0u && localNodeId > 0) --localNodeId;

				if (localNodeId <= 1)
				{
					//did not find a mathcing node in the second graph, invalidate the subgraph
					//invalidateSubgraph(subgraphStartLocation);
					return false;
				}

				outNodeIds[subgraphStartLocation + localNodeId] = outNodeIds[subgraphStartLocation + localNodeId] + 1;
				outBorderNodeFlags[subgraphStartLocation + localNodeId] = 0u;
				--localNodeId;//cancel out post-increment from the loop
			}
		}//end for all nodes in the cut

     	//double check selected matching nodes
		bool foundMismatch = false;
		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize && !foundMismatch; ++localNodeId)
		{
			if (inBorderNodeFlags[subgraphStartLocation + localNodeId] == 0u)
				continue;

			unsigned int inNodeId = inNodeIds[subgraphStartLocation + localNodeId];
			unsigned int inNodeType = inNodeTypes[inNodeId];
			unsigned int outNodeId = outNodeIds[subgraphStartLocation + localNodeId];
			unsigned int outNodeType = outNodeTypes[outNodeId];
			if (inNodeType != outNodeType)
			{
				foundMismatch = true;
				break;
			}

			for (unsigned int pairingNodeId = 0; pairingNodeId < subgraphSize; ++pairingNodeId)
			{
				if (inBorderNodeFlags[subgraphStartLocation + pairingNodeId] == 0u)
					continue;
				if (pairingNodeId == localNodeId)
					continue;
				unsigned int inPairNodeId = inNodeIds[subgraphStartLocation + pairingNodeId];
				unsigned int outPairNodeId = outNodeIds[subgraphStartLocation + pairingNodeId];
				const float targetDist = inDistMatrix[inNodeId + graphSize1 * inPairNodeId];
				const float currentDist = outDistMatrix[outNodeId + graphSize2 * outPairNodeId];
				if (fabsf(currentDist - targetDist) > spatialTolerance * outNodeSizes[outPairNodeId])
				{
					foundMismatch = true;
					break;
				}
			}//end for each other node in the cut
		}//end for each node in the cut

		if (!foundMismatch)
		{
			outValidSubgraphFlags[aId] = 1u;
			return true;
		}

		return false;
	}


	__host__ __device__	void operator()(const size_t& aId_s)
	{
		unsigned int aId = (unsigned int)aId_s;
		//unsigned int subgraphSeedNodeId = subgraphsPerSeedNode == 0u ? aId : aId / subgraphsPerSeedNode;
		//unsigned int subgraphOffset = subgraphsPerSeedNode == 0u ? 0u : aId % subgraphsPerSeedNode;
		//unsigned int subgraphStartLocation = subgraphOffset * subgraphSize + subgraphSeedNodeId * subgraphsPerSeedNode * subgraphSize;
		
		KISSRandomNumberGenerator genRand(
			3643u + aId * 4154207u * graphSize2 + graphSize2,
			1761919u + aId * 2746753u * graphSize1,
			331801u + aId,
			10499029u);

		unsigned int offset = (unsigned int)(genRand() * (float)graphSize2);
		bool success = randomMatchingOperator(aId, offset);
		//if(!success)
		//	success = randomMatchingOperator(aId, offset / 2u);
		//if(!success)
		//	randomMatchingOperator(aId, offset / 4u);
		if(!success)
			randomMatchingOperator(aId, 0u);

	}


};

class TransformationEstimator
{
public:
	unsigned int subgraphSize;

	thrust::device_ptr<float3> positions1;
	thrust::device_ptr<float3> positions2;

	thrust::device_ptr<unsigned int> nodeIds1;
	thrust::device_ptr<unsigned int> borderNodeFlags;
	thrust::device_ptr<unsigned int> nodeIds2;

	thrust::device_ptr<unsigned int> outValidSubgraphFlags;

	thrust::device_ptr<float3> outTranslation1;
	thrust::device_ptr<float3> outTranslation2;
	thrust::device_ptr<float> tmpCovMatrix;
	thrust::device_ptr<float> tmpDiagonalW;
	thrust::device_ptr<float> tmpMatrixV;
	thrust::device_ptr<float> tmpVecRV;
	thrust::device_ptr<quaternion4f> outRotation2;


	TransformationEstimator(
		unsigned int aSampleSize,
		thrust::device_ptr<float3> aPositions1,
		thrust::device_ptr<float3> aPositions2,
		thrust::device_ptr<unsigned int> inIds,
		thrust::device_ptr<unsigned int> inFlags,
		thrust::device_ptr<unsigned int> outIds,
		thrust::device_ptr<unsigned int> outSubgraphFlags,
		thrust::device_ptr<float3> aTranslation1,
		thrust::device_ptr<float3> aTranslation2,
		thrust::device_ptr<float> aCovMatrix,
		thrust::device_ptr<float> aDiagonalW,
		thrust::device_ptr<float> aMatrixV,
		thrust::device_ptr<float> aVecRV,
		thrust::device_ptr<quaternion4f> aOutRot
	) : subgraphSize(aSampleSize),
		positions1(aPositions1),
		positions2(aPositions2),
		nodeIds1(inIds),
		borderNodeFlags(inFlags),
		nodeIds2(outIds),
		outValidSubgraphFlags(outSubgraphFlags),
		outTranslation1(aTranslation1),
		outTranslation2(aTranslation2),
		tmpCovMatrix(aCovMatrix),
		tmpDiagonalW(aDiagonalW),
		tmpMatrixV(aMatrixV),
		tmpVecRV(aVecRV),
		outRotation2(aOutRot)
	{}

	__host__ __device__	void operator()(const size_t& aId_s)
	{
		unsigned int aId = (unsigned int)aId_s;
		if (outValidSubgraphFlags[aId] == 0u)
			return;

		unsigned int subgraphStartLocation = aId * subgraphSize;// (aId % 32) * subgraphSize;

		//Compute the means of the border node locations
		float3 center1 = make_float3(0.f, 0.f, 0.f);
		float3 center2 = make_float3(0.f, 0.f, 0.f);
		float numPoints = 0.f;
		for (unsigned int i = 0u; i < subgraphSize; ++i)
		{
			if (borderNodeFlags[subgraphStartLocation + i] != 0u)
			{
				center1 += positions1[nodeIds1[subgraphStartLocation + i]];
				center2 += positions2[nodeIds2[subgraphStartLocation + i]];
				numPoints += 1.f;
			}
		}
		center1 /= numPoints;
		center2 /= numPoints;

		//Compute covariance matrix
		float* covMat = thrust::raw_pointer_cast(tmpCovMatrix + aId * 3 * 3);
		for (unsigned int i = 0u; i < subgraphSize; ++i)
		{
			if (borderNodeFlags[subgraphStartLocation + i] != 0u)
			{
				float3 vec1 = positions1[nodeIds1[subgraphStartLocation + i]] - center1;
				float3 vec2 = positions2[nodeIds2[subgraphStartLocation + i]] - center2;

				covMat[0 * 3 + 0] += vec2.x * vec1.x;
				covMat[1 * 3 + 0] += vec2.y * vec1.x;
				covMat[2 * 3 + 0] += vec2.z * vec1.x;

				covMat[0 * 3 + 1] += vec2.x * vec1.y;
				covMat[1 * 3 + 1] += vec2.y * vec1.y;
				covMat[2 * 3 + 1] += vec2.z * vec1.y;

				covMat[0 * 3 + 2] += vec2.x * vec1.z;
				covMat[1 * 3 + 2] += vec2.y * vec1.z;
				covMat[2 * 3 + 2] += vec2.z * vec1.z;
			}
		}
		//Singular Value Decomposition
		float* diag = thrust::raw_pointer_cast(tmpDiagonalW + aId * 3);
		float* vMat = thrust::raw_pointer_cast(tmpMatrixV + aId * 3 * 3);
		float* tmp = thrust::raw_pointer_cast(tmpVecRV + aId * 3);

		svd::svdcmp(covMat, 3, 3, diag, vMat, tmp);

		//Rotation is V * transpose(U)		
		for (unsigned int row = 0; row < 3; ++row)
		{
			for (unsigned int col = 0; col < 3; ++col)
			{
				tmp[col] =
					vMat[row * 3 + 0] * covMat[col * 3 + 0] +
					vMat[row * 3 + 1] * covMat[col * 3 + 1] +
					vMat[row * 3 + 2] * covMat[col * 3 + 2];
			}
			vMat[row * 3 + 0] = tmp[0];
			vMat[row * 3 + 1] = tmp[1];
			vMat[row * 3 + 2] = tmp[2];
		}


		float rotDet = determinant(
			vMat[0], vMat[3], vMat[6],
			vMat[1], vMat[4], vMat[7],
			vMat[2], vMat[5], vMat[8]
		);

		if (rotDet < 0.f)
		{
			vMat[6] = -vMat[6];
			vMat[7] = -vMat[7];
			vMat[8] = -vMat[8];
			rotDet = -rotDet;
		}

		if (fabsf(rotDet - 1.f)> EPS)
			outValidSubgraphFlags[aId] = 0u;


		quaternion4f rotation(
			vMat[0], vMat[3], vMat[6],
			vMat[1], vMat[4], vMat[7],
			vMat[2], vMat[5], vMat[8]
		);
		outTranslation1[aId] = center1;
		outTranslation2[aId] = center2;
		outRotation2[aId] = rotation;
	}

};


__host__ std::string VariationGenerator::operator()(const char * aFilePath1, const char * aFilePath2,
	WFObject & aObj1, WFObject & aObj2, Graph & aGraph1, Graph & aGraph2, float aRelativeThreshold)
{
	cudastd::timer timer;
	cudastd::timer intermTimer;

	if (aGraph1.numNodes() < 9u || aGraph2.numNodes() < 9u)
		return "";

	thrust::host_vector<float3> objCenters1;
	thrust::host_vector<float> objSizes1;

	ObjectCenterExporter()(aObj1, objCenters1, objSizes1, 0.3333f);

	thrust::host_vector<float3> objCenters2;
	thrust::host_vector<float> objSizes2;

	ObjectCenterExporter()(aObj2, objCenters2, objSizes2, 0.3333f);

	thrust::device_vector<float3> centersDevice1(objCenters1);
	thrust::device_vector<float> pairwiseDistMatrix1(objCenters1.size() * objCenters1.size());
	thrust::counting_iterator<size_t> first(0u);
	thrust::counting_iterator<size_t> last1(pairwiseDistMatrix1.size());
	DistanceMatrixWriter writeDistances1(objCenters1.size(), pairwiseDistMatrix1.data(), centersDevice1.data());
	thrust::for_each(first, last1, writeDistances1);

	thrust::host_vector<unsigned int> nodeTypes1Host(aGraph1.numNodes(), (unsigned int)aObj1.materials.size());
	for (size_t nodeId = 0; nodeId < aObj1.objects.size(); ++nodeId)
	{
		size_t faceId = aObj1.objects[nodeId].x;
		size_t materialId = aObj1.faces[faceId].material;
		nodeTypes1Host[nodeId] = (unsigned int)materialId;
	}
	thrust::device_vector<unsigned int> nodeTypes1(nodeTypes1Host);

	thrust::device_vector<float3> centersDevice2(objCenters2);
	thrust::device_vector<float> pairwiseDistMatrix2(objCenters2.size() * objCenters2.size());
	thrust::counting_iterator<size_t> last2(pairwiseDistMatrix2.size());
	DistanceMatrixWriter writeDistances2(objCenters2.size(), pairwiseDistMatrix2.data(), centersDevice2.data());
	thrust::for_each(first, last2, writeDistances2);

	thrust::host_vector<unsigned int> nodeTypes2Host(aGraph2.numNodes(), (unsigned int)aObj2.materials.size());
	for (size_t nodeId = 0u; nodeId < aObj2.objects.size(); ++nodeId)
	{
		size_t faceId = aObj2.objects[nodeId].x;
		size_t materialId = aObj2.faces[faceId].material;
		nodeTypes2Host[nodeId] = (unsigned int)materialId;
	}
	thrust::device_vector<unsigned int> nodeTypes2(nodeTypes2Host);

	float3 minBound, maxBound;
	ObjectBoundsExporter()(aObj1, minBound, maxBound);
	const float boundsDiagonal = len(maxBound - minBound);
	const float spatialTolerance = boundsDiagonal * std::max(aRelativeThreshold, 0.02f);
	//const float spatialTolerance = 30.f * (aRelativeThreshold + 0.03f);


	std::string result = "";

	initTime = intermTimer.get();
	intermTimer.start();

	samplingTime = matchingTime = svdTime = matchingTime = 0.f;
	histTime = transformTime = collisionTime = exportTime = conversionTime = 0.f;
	histoChecks = matchingCuts = matchingCutsAndTs = histoChecksPassed = 0u;

	std::vector<NodeTypeHistogram> variatioHistograms;
	variatioHistograms.push_back(NodeTypeHistogram(nodeTypes1Host));
	variatioHistograms.push_back(NodeTypeHistogram(nodeTypes2Host));

	thrust::host_vector<unsigned int> graph2Intervals(aGraph2.intervals);
	thrust::host_vector<unsigned int> graph2NbrIds(aGraph2.adjacencyVals);
	thrust::host_vector<unsigned int> graph1Intervals(aGraph1.intervals);
	thrust::host_vector<unsigned int> graph1NbrIds(aGraph1.adjacencyVals);

	thrust::device_vector<float> objSizes1Device(objSizes1);
	thrust::device_vector<float> objSizes2Device(objSizes2);

	GrammarCheck grammarCheck;
	grammarCheck.init(graph2Intervals, graph2NbrIds, nodeTypes2Host);
	grammarCheck.init(graph1Intervals, graph1NbrIds, nodeTypes1Host);

	numVariations = 0u;

	const unsigned int numSubgraphSamples = 32u * (unsigned int)aGraph1.numNodes();// std::max(aGraph1.numNodes(), aGraph2.numNodes());
	//const unsigned int subgraphSampleSize = (unsigned int)objCenters1.size() / 2u;

	for (unsigned int subgraphSampleSize = 4u;// (unsigned int)std::min(aGraph1.numNodes(), aGraph2.numNodes()) / 4u;
		subgraphSampleSize < (unsigned int) 3u * aGraph1.numNodes() / 4u;
		subgraphSampleSize++)
	{
		std::cout << "Subgraph sample size: " << subgraphSampleSize << " / " << 3u * aGraph1.numNodes() / 4u <<"\r";

		if (subgraphSampleSize < 3)
			continue;

		thrust::device_vector<unsigned int> subgraphNodeIds1(numSubgraphSamples * subgraphSampleSize);
		thrust::device_vector<unsigned int> subgraphBorderFlags1(numSubgraphSamples * subgraphSampleSize);

		thrust::device_vector<unsigned int> subgraphNodeIds2(numSubgraphSamples * subgraphSampleSize);
		thrust::device_vector<unsigned int> subgraphBorderFlags2(numSubgraphSamples * subgraphSampleSize);

		SubgraphInitializer initSubgraphSamples(
			(unsigned int)objCenters1.size(),
			subgraphSampleSize,
			numSubgraphSamples,
			aGraph1.intervals.data(),
			aGraph1.adjacencyVals.data(),
			subgraphNodeIds1.data(),
			subgraphBorderFlags1.data());

		thrust::counting_iterator<size_t> lastSubgraph(numSubgraphSamples);
		thrust::for_each(first, lastSubgraph, initSubgraphSamples);

		samplingTime += intermTimer.get();
		intermTimer.start();

		//#ifdef _DEBUG
		//	outputDeviceVector("Subgraph node ids     1: ", subgraphNodeIds1);
		//	outputDeviceVector("Subgraph border flags 1: ", subgraphBorderFlags1);
		//#endif

		///////////////////////////////////////////////////////////////////////////////////
		//Find matching cuts in both sub-graphs

		thrust::device_vector<unsigned int> validSubgraphFlags(numSubgraphSamples, 0u);

		CutMatching matchCuts(
			(unsigned int)objCenters1.size(),
			(unsigned int)objCenters2.size(),
			subgraphSampleSize,
			numSubgraphSamples,
			spatialTolerance,
			nodeTypes1.data(),
			subgraphNodeIds1.data(),
			subgraphBorderFlags1.data(),
			pairwiseDistMatrix1.data(),
			objSizes1Device.data(),
			nodeTypes2.data(),
			subgraphNodeIds2.data(),
			subgraphBorderFlags2.data(),
			pairwiseDistMatrix2.data(),
			objSizes2Device.data(),
			validSubgraphFlags.data()
		);

		//thrust::counting_iterator<size_t> lastSubgraphDbg(4);
		thrust::for_each(first, lastSubgraph, matchCuts);

		matchingCuts += thrust::count(validSubgraphFlags.begin(), validSubgraphFlags.end(), 1u);

		matchingTime += intermTimer.get();
		intermTimer.start();

		//#ifdef _DEBUG
		//	outputDeviceVector("Subgraph node ids     2: ", subgraphNodeIds2);
		//	outputDeviceVector("Subgraph border flags 2: ", subgraphBorderFlags2);
		//	outputDeviceVector("Valid subgraph flags   : ", validSubgraphFlags);
		//#endif


		///////////////////////////////////////////////////////////////////////////////////
		//Find correspondence transformation between both sub-graphs
		thrust::device_vector<float3> outTranslation1(numSubgraphSamples);
		thrust::device_vector<float3> outTranslation2(numSubgraphSamples);
		thrust::device_vector<float> tmpCovMatrix(numSubgraphSamples * 3 * 3, 0.f);
		thrust::device_vector<float> tmpDiagonalW(numSubgraphSamples * 3);
		thrust::device_vector<float> tmpMatrixV(numSubgraphSamples * 3 * 3);
		thrust::device_vector<float> tmpVecRV(numSubgraphSamples * 3);
		thrust::device_vector<quaternion4f> outRotation2(numSubgraphSamples);

		TransformationEstimator estimateT(
			subgraphSampleSize,
			centersDevice1.data(),
			centersDevice2.data(),
			subgraphNodeIds1.data(),
			subgraphBorderFlags1.data(),
			subgraphNodeIds2.data(),
			validSubgraphFlags.data(),
			outTranslation1.data(),
			outTranslation2.data(),
			tmpCovMatrix.data(),
			tmpDiagonalW.data(),
			tmpMatrixV.data(),
			tmpVecRV.data(),
			outRotation2.data()
		);

		thrust::for_each(first, lastSubgraph, estimateT);

		matchingCutsAndTs += thrust::count(validSubgraphFlags.begin(), validSubgraphFlags.end(), 1u);

		svdTime += intermTimer.get();
		intermTimer.start();

		///////////////////////////////////////////////////////////////////////////////////
		//Copy back to host
		thrust::host_vector<unsigned int> subgraphNodeIdsHost1(subgraphNodeIds1);
		thrust::host_vector<unsigned int> subgraphBorderFlagsHost1(subgraphBorderFlags1);

		thrust::host_vector<unsigned int> subgraphNodeIdsHost2(subgraphNodeIds2);
		thrust::host_vector<unsigned int> subgraphBorderFlagsHost2(subgraphBorderFlags2);

		thrust::host_vector<unsigned int> validSubgraphFlagsHost(validSubgraphFlags);

		thrust::host_vector<float3> outTranslation1Host(outTranslation1);
		thrust::host_vector<float3> outTranslation2Host(outTranslation2);
		thrust::host_vector<quaternion4f> outRotation2Host(outRotation2);

		unsigned int graphSize1 = (unsigned int)objCenters1.size();
		unsigned int graphSize2 = (unsigned int)objCenters2.size();

		GraphToStringConverter convertToStr;
		CollisionGraphExporter graphExporter;
		WFObjectFileExporter   objExporter;

		cpyBackTime += intermTimer.get();
		intermTimer.start();

		for (unsigned int subgraphId = 0u; subgraphId < numSubgraphSamples; ++subgraphId)
		{
			intermTimer.start();

			if (validSubgraphFlagsHost[subgraphId] != 1u)
				continue;

			thrust::host_vector<unsigned int> completeSubgraphFlags2(graphSize2, 0u);
			std::vector<unsigned int> nodeStack;
			unsigned int subgraph2Size = 0u;
			unsigned int complementSize = 0u;
			thrust::host_vector<unsigned int>::iterator subgraphNodeIdsHost1Begin = subgraphNodeIdsHost1.begin() + subgraphId * subgraphSampleSize;
			thrust::host_vector<unsigned int>::iterator subgraphNodeIdsHost2Begin = subgraphNodeIdsHost2.begin() + subgraphId * subgraphSampleSize;
			thrust::host_vector<unsigned int>::iterator subgraphBorderFlagsHost1Begin = subgraphBorderFlagsHost1.begin() + subgraphId * subgraphSampleSize;
			thrust::host_vector<unsigned int>::iterator subgraphBorderFlagsHost2Begin = subgraphBorderFlagsHost2.begin() + subgraphId * subgraphSampleSize;

			//initialize flags at graph cut - 2 -> outside node, 1 -> border node
			for (unsigned int i = 0u; i < subgraphSampleSize; ++i)
			{
				if (*(subgraphBorderFlagsHost2Begin + i) != 0u)
					completeSubgraphFlags2[*(subgraphNodeIdsHost2Begin + i)] = *(subgraphBorderFlagsHost2Begin + i);
				if (*(subgraphBorderFlagsHost2Begin + i) == 1u)
				{
					++subgraph2Size;
					nodeStack.push_back(*(subgraphNodeIdsHost2Begin + i));
				}
				if (*(subgraphBorderFlagsHost2Begin + i) == 2u)
					++complementSize;
			}
			//region grow from each border node
			while (!nodeStack.empty())
			{
				unsigned int nodeId = nodeStack.back();
				nodeStack.pop_back();
				for (unsigned int nbr = graph2Intervals[nodeId]; nbr < graph2Intervals[nodeId + 1]; ++nbr)
				{
					unsigned int nbrId = graph2NbrIds[nbr];
					if (completeSubgraphFlags2[nbrId] == 2u || completeSubgraphFlags2[nbrId] == 1u)
						continue;
					completeSubgraphFlags2[nbrId] = 1u;
					nodeStack.push_back(nbrId);
					++subgraph2Size;
				}
			}
			//check validity
			if (subgraph2Size + complementSize >= graphSize2)
				continue; //should not happen
			unsigned int subgraph1Size = 0u;
			thrust::host_vector<unsigned int> completeSubgraphFlags1(graphSize1, 0u);
			for (unsigned int i = 0u; i < subgraphSampleSize; ++i)
			{
				if (*(subgraphBorderFlagsHost1Begin + i) == 0u || *(subgraphBorderFlagsHost1Begin + i) == 2u)
				{
					completeSubgraphFlags1[*(subgraphNodeIdsHost1Begin + i)] = 1u;
					++subgraph1Size;
				}
			}

			///////////////////////////////////////////////////////////////////////////////////
			//discard variations with repeating node type histograms
			variatioHistograms.push_back(NodeTypeHistogram(aObj1.materials.size()));
			for (auto inTypeIt1 = nodeTypes1Host.begin(); inTypeIt1 != nodeTypes1Host.end(); ++inTypeIt1)
			{
				if (completeSubgraphFlags1[inTypeIt1 - nodeTypes1Host.begin()] == 1u)
				{
					variatioHistograms.back().typeCounts[*inTypeIt1]++;
				}
			}
			for (auto inTypeIt2 = nodeTypes2Host.begin(); inTypeIt2 != nodeTypes2Host.end(); ++inTypeIt2)
			{
				if (completeSubgraphFlags2[inTypeIt2 - nodeTypes2Host.begin()] == 1u)
				{
					variatioHistograms.back().typeCounts[*inTypeIt2]++;
				}
			}

			bool repeatedHistogram = false;
			for (size_t hid = 0u; hid < variatioHistograms.size() - 1 && !repeatedHistogram; ++hid)
			{
				++histoChecks;
				if (variatioHistograms.back() == variatioHistograms[hid])
					repeatedHistogram = true;
			}


			histTime += intermTimer.get();
			intermTimer.start();

			if (repeatedHistogram)
			{
				variatioHistograms.pop_back();
				continue;
			}

			++histoChecksPassed;
			////////////////////////////////////////////////////////////////////////////////////////

			for (unsigned int i = 0u; i < graphSize2; ++i)
			{
				if (completeSubgraphFlags2[i] == 2u)
					completeSubgraphFlags2[i] = 0u;
			}

			//graphExporter.exportSubGraph(aFilePath1, aObj1, aGraph1, numVariations, completeSubgraphFlags1);
			//graphExporter.exportSubGraph(aFilePath2, aObj2, aGraph2, numVariations, completeSubgraphFlags2);

			///////////////////////////////////////////////////////////////////////////////////
			//Create the variation by merging the subsets of aObj1 and aObj2
			float3 translation1 = outTranslation1Host[subgraphId];
			float3 translation2 = outTranslation2Host[subgraphId];
			quaternion4f rotation2 = outRotation2Host[subgraphId];
			WFObject variation = WFObjectMerger()(aObj1, translation1, aObj2, translation2, rotation2, completeSubgraphFlags1, completeSubgraphFlags2);
			///////////////////////////////////////////////////////////////////////////////////
			transformTime += intermTimer.get();
			intermTimer.start();
			///////////////////////////////////////////////////////////////////////////////////
			//Compute the collision graph for the variation
			CollisionDetector detector;
			Graph variationGraph = detector.computeCollisionGraph(variation, aRelativeThreshold);
			///////////////////////////////////////////////////////////////////////////////////
			collisionTime += intermTimer.get();
			intermTimer.start();
			///////////////////////////////////////////////////////////////////////////////////
			//Check that the variation graph is valid
			thrust::host_vector<unsigned int> nodeTypesVariation(variationGraph.numNodes());
			for (size_t nodeId = 0; nodeId < variationGraph.numNodes(); ++nodeId)
			{
				size_t faceId = variation.objects[nodeId].x;
				size_t materialId = variation.faces[faceId].material;
				nodeTypesVariation[nodeId] = (unsigned int)materialId;
			}
			thrust::host_vector<unsigned int> hostIntervals(variationGraph.intervals);
			thrust::host_vector<unsigned int> hostNbrIds(variationGraph.adjacencyVals);
			if (!grammarCheck.check(hostIntervals, hostNbrIds, nodeTypesVariation))
			{
				//variationGraph = detector.computeCollisionGraph(variation, std::max(aRelativeThreshold, 0.02f));
				//hostIntervals = variationGraph.intervals;
				//hostNbrIds = variationGraph.adjacencyVals;
				//if (!grammarCheck.check(hostIntervals, hostNbrIds, nodeTypesVariation))
				//{
					variatioHistograms.pop_back();
					continue;
				//}

			}
			///////////////////////////////////////////////////////////////////////////////////
			++numVariations;

			if (writeVariationGraphs || writeVariations)
			{
				std::string fileName1(aFilePath1);
				if (fileName1.find_last_of("/\\") == std::string::npos)
					fileName1 = fileName1.substr(0, fileName1.size() - 5);
				else
					fileName1 = fileName1.substr(fileName1.find_last_of("/\\") + 1, fileName1.size() - fileName1.find_last_of("/\\") - 5);

				std::string fileName2(aFilePath2);
				if (fileName2.find_last_of("/\\") == std::string::npos)
					fileName2 = fileName2.substr(0, fileName2.size() - 5);
				else
					fileName2 = fileName2.substr(fileName2.find_last_of("/\\") + 1, fileName2.size() - fileName2.find_last_of("/\\") - 5);

				std::string objDir = getDirName(aFilePath2);
				std::string variationFilePath = objDir + fileName1 + "_" + fileName2 + "_" + itoa((int)numVariations);
				if (writeVariations)
					objExporter(variation, variationFilePath.c_str());
				
				if (writeVariationGraphs)
					graphExporter.exportCollisionGraph((variationFilePath + ".obj").c_str(), variation, variationGraph);
			}

			exportTime = intermTimer.get();
			intermTimer.start();

			std::string variationStrings = convertToStr(variation, variationGraph);
			if (!multiString)
				variationStrings = variationStrings.substr(0u, variationStrings.find_first_of("\n"));
			result.append(variationStrings);

			conversionTime += intermTimer.get();
		}//end for subgraph samples

	}//end for subgraph size

	std::cout << "Total subgraph samples: " << numSubgraphSamples * (unsigned int)3u * aGraph1.numNodes() / 4 << "\n";

	totalTime = timer.get();

	intermTimer.cleanup();
	timer.cleanup();

	return result;
}


__host__ void VariationGenerator::stats()
{
	size_t miliseconds = (size_t)totalTime % 1000u;
	size_t minutes = (size_t)totalTime / 60000;
	size_t seconds = ((size_t)totalTime % 60000) / 1000;

	std::cerr << "Created "<< numVariations <<" variations in " << totalTime << "ms "
		<< minutes << ":" << seconds << ":" << miliseconds << " (min:sec:ms)\n";
	std::cerr << "Matching subgraph cuts   : " << matchingCuts << "\n";
	std::cerr << "Matching transformations : " << matchingCutsAndTs << "\n";
	std::cerr << "New histograms           : " << histoChecksPassed << "\n";
	std::cerr << "Grammar checks passed    : " << numVariations << "\n";
	std::cerr << "-------------------------------------\n";
	std::cerr << "Initialization in      " << initTime << "ms\n";
	std::cerr << "Subgraph sampling in   " << samplingTime << "ms\n";
	std::cerr << "Graph cut matching in  " << matchingTime << "ms\n";
	std::cerr << "SVD in                 " << svdTime << "ms\n";
	std::cerr << "Mem transfer in        " << cpyBackTime << "ms\n";
	std::cerr << "Histogram check  in    " << histTime << "ms (performed   " << histoChecks << " checks)\n";
	std::cerr << "Obj transformation in  " << transformTime << "ms\n";
	std::cerr << "Collision detection in " << collisionTime << "ms\n";
	std::cerr << "File export in         " << exportTime << "ms\n";
	std::cerr << "String conversion      " << conversionTime << "ms\n";
}