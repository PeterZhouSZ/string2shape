#include "hip/hip_runtime.h"
#include "pch.h"
#include "VariationGenerator.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/scan.h>

#include "Algebra.h"
#include "SVD.h"
#include "WFObjUtils.h"
#include "Graph2String.h"
#include "CollisionDetector.h"
#include "CollisionGraphExporter.h"

#include "DebugUtils.h"
#include "Timer.h"


class DistanceMatrixWriter
{
public:
	size_t stride;
	thrust::device_ptr<float> matrix;
	thrust::device_ptr<float3> positions;


	DistanceMatrixWriter(
		size_t aStride,
		thrust::device_ptr<float> aMatrix,
		thrust::device_ptr<float3> aPositions
	) :stride(aStride), matrix(aMatrix), positions(aPositions)
	{}

	__host__ __device__	void operator()(const size_t& aId)
	{
		const size_t myRowId = aId % stride;
		const size_t myColId = aId / stride;
		const float3 objCenter1 = positions[myColId];
		const float3 objCenter2 = positions[myRowId];
		matrix[myColId + myRowId * stride] = len(objCenter1 - objCenter2);
	}

};

class KISSRandomNumberGenerator
{
public:
	uint data[4];
	//data[0],
	//data[1],//must be zero
	//data[2],
	//data[3]; //doesn't need to be re-seeded but must be < 698769069

	__host__ __device__ KISSRandomNumberGenerator(
		const uint aX = 123456789u,
		const uint aY = 362436069u,
		const uint aZ = 521288629u,
		const uint aW = 416191069u)
	{
		data[0] = (aX); data[1] = (aY); data[2] = (aZ); data[3] = (aW);
	}

	__host__ __device__ float operator()()
	{
		data[2] = (36969 * (data[2] & 65535) + (data[2] >> 16)) << 16;
		data[3] = 18000 * (data[3] & 65535) + (data[3] >> 16) & 65535;
		data[0] = 69069 * data[0] + 1234567;
		data[1] = (data[1] = (data[1] = data[1] ^ (data[1] << 17)) ^ (data[1] >> 13)) ^ (data[1] << 5);
		return ((data[2] + data[3]) ^ data[0] + data[1]) * 2.328306E-10f;
	}
};

class SubgraphInitializer
{
public:
	unsigned int graphSize;
	unsigned int subgraphSize;
	unsigned int numSubgraphs;
	unsigned int subgraphsPerSeedNode;
	//graph edges
	thrust::device_ptr<unsigned int> adjIntervals;
	thrust::device_ptr<unsigned int> neighborIds;

	thrust::device_ptr<unsigned int> outNodeIds;
	thrust::device_ptr<unsigned int> outBorderNodeFlags;

	SubgraphInitializer(
		unsigned int aGraphSize,
		unsigned int aSampleSize,
		unsigned int aNumSamples,
		thrust::device_ptr<unsigned int> aIntervals,
		thrust::device_ptr<unsigned int> aNeighborIds,
		thrust::device_ptr<unsigned int> outIds,
		thrust::device_ptr<unsigned int> outFlags
		) : graphSize(aGraphSize),
		subgraphSize(aSampleSize),
		numSubgraphs(aNumSamples),
		subgraphsPerSeedNode(aNumSamples / aGraphSize),
		adjIntervals(aIntervals),
		neighborIds(aNeighborIds),
		outNodeIds(outIds),
		outBorderNodeFlags(outFlags)
	{}

	__host__ __device__	void operator()(const size_t& aId_s)
	{
		unsigned int aId = (unsigned int)aId_s;

		KISSRandomNumberGenerator genRand(
			3643u + aId * 4154207u * subgraphsPerSeedNode + numSubgraphs,
			1761919u + aId * 2746753u * subgraphsPerSeedNode ,
			331801u + aId,
			10499029u);

		unsigned int subgraphSeedNodeId = subgraphsPerSeedNode == 0u ? aId : aId / subgraphsPerSeedNode;
		unsigned int subgraphOffset = subgraphsPerSeedNode == 0u ? 0u : aId % subgraphsPerSeedNode;
		unsigned int subgraphStartLocation = subgraphOffset * subgraphSize + subgraphSeedNodeId * subgraphsPerSeedNode * subgraphSize;

		outNodeIds[subgraphStartLocation] = subgraphSeedNodeId;

		unsigned int currentSize = 1u;
		unsigned int currentDepth = 0u;
		unsigned int currentSubgraphNodeId = 0u;

		//compute subgraph
		while (currentSize < subgraphSize && currentSize < graphSize)
		{
			unsigned int neighborCount = 0u;
			for (unsigned int localNodeId = currentSubgraphNodeId; localNodeId < currentSize; ++localNodeId)
			{
				unsigned int nodeId = outNodeIds[subgraphStartLocation + localNodeId];
				for (unsigned int localNeighborId = adjIntervals[nodeId]; localNeighborId < adjIntervals[nodeId + 1u]; ++localNeighborId)
				{
					unsigned int neighborId = neighborIds[localNeighborId];
					bool alreadyIncluded = false;
					for (unsigned int previousNodeId = 0u; previousNodeId < currentSize + neighborCount; ++previousNodeId)
					{						
						if (outNodeIds[subgraphStartLocation + previousNodeId] == neighborId)
						{
							alreadyIncluded = true;
							break;
						}
					}

					if (!alreadyIncluded && neighborCount + currentSize < subgraphSize) //add to subgraph
					{
						outNodeIds[subgraphStartLocation + neighborCount + currentSize] = neighborId;
						neighborCount++;
					}
					else if (!alreadyIncluded && genRand() < 0.5)//replace a random node with the same depth
					{
						unsigned int randLocation = (int)(genRand() * (float)neighborCount);
						outNodeIds[subgraphStartLocation + randLocation + currentSize] = neighborId;
					}
				}

			}

			currentSubgraphNodeId = currentSize;
			currentDepth++;
			currentSize += neighborCount;
		}
		//compute subgraph interior and border
		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
		{
			unsigned int nodeId = outNodeIds[subgraphStartLocation + localNodeId];
			bool allNeighborsIncluded = true;
			for (unsigned int localNeighborId = adjIntervals[nodeId]; localNeighborId < adjIntervals[nodeId + 1u]; ++localNeighborId)
			{
				unsigned int neighborId = neighborIds[localNeighborId];
				bool alreadyIncluded = false;
				for (unsigned int previousNodeId = 0u; previousNodeId < subgraphSize; ++previousNodeId)
				{
					if (outNodeIds[subgraphStartLocation + previousNodeId] == neighborId)
					{
						alreadyIncluded = true;
						break;
					}
				}
				if (!alreadyIncluded)
				{
					allNeighborsIncluded = false;
					break;
				}
			}

			if (allNeighborsIncluded)
			{
				outBorderNodeFlags[subgraphStartLocation + localNodeId] = 0u;//mark as interior
			}
			else
			{
				outBorderNodeFlags[subgraphStartLocation + localNodeId] = 1u;//mark as border node
			}
		}

		//compute inner border
		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
		{
			unsigned int nodeId = outNodeIds[subgraphStartLocation + localNodeId];
			bool nextToBorder = false;
			for (unsigned int localNeighborId = adjIntervals[nodeId]; localNeighborId < adjIntervals[nodeId + 1u] && !nextToBorder; ++localNeighborId)
			{
				unsigned int neighborId = neighborIds[localNeighborId];
				for (unsigned int previousNodeId = 0u; previousNodeId < subgraphSize; ++previousNodeId)
				{
					if (outNodeIds[subgraphStartLocation + previousNodeId] == neighborId &&
						outBorderNodeFlags[subgraphStartLocation + previousNodeId] == 1u)
					{
						outBorderNodeFlags[subgraphStartLocation + localNodeId] = 2u;//mark as inner border node
						nextToBorder = true;
						break;
					}
				}
			}

		}
	}

};

class CutMatching
{
public:
	unsigned int graphSize1;
	unsigned int graphSize2;
	unsigned int subgraphSize;
	unsigned int numSubgraphs;
	unsigned int subgraphsPerSeedNode;
	float spatialTolerance;
	//graph 1
	//thrust::device_ptr<unsigned int> inIntervals;
	//thrust::device_ptr<unsigned int> inNeighborIds;
	thrust::device_ptr<unsigned int> inNodeTypes;
	//subrgaphs 1
	thrust::device_ptr<unsigned int> inNodeIds;
	thrust::device_ptr<unsigned int> inBorderNodeFlags;
	//node-node distances 1
	thrust::device_ptr<float>        inDistMatrix;
	//graph 2
	thrust::device_ptr<unsigned int> outNodeTypes;
	//subrgaphs 2
	thrust::device_ptr<unsigned int> outNodeIds;
	thrust::device_ptr<unsigned int> outBorderNodeFlags;
	//node-node distances 2
	thrust::device_ptr<float>        outDistMatrix;

	thrust::device_ptr<unsigned int> outValidSubgraphFlags;

	CutMatching(
		unsigned int aGraphSize1,
		unsigned int aGraphSize2,
		unsigned int aSampleSize,
		unsigned int aNumSamples,
		float aSpatialTolerance,
		//thrust::device_ptr<unsigned int> aIntervals,
		//thrust::device_ptr<unsigned int> aNeighborIds,
		thrust::device_ptr<unsigned int> inTypes,
		thrust::device_ptr<unsigned int> inIds,
		thrust::device_ptr<unsigned int> inFlags,
		thrust::device_ptr<float> inMatrix,
		thrust::device_ptr<unsigned int> outTypes,
		thrust::device_ptr<unsigned int> outIds,
		thrust::device_ptr<unsigned int> outFlags,
		thrust::device_ptr<float> outMatrix,
		thrust::device_ptr<unsigned int> outSubgraphFlags
	) : graphSize1(aGraphSize1),
		graphSize2(aGraphSize2),
		subgraphSize(aSampleSize),
		numSubgraphs(aNumSamples),
		subgraphsPerSeedNode(aNumSamples / aGraphSize1),
		spatialTolerance(aSpatialTolerance),
		//inIntervals(inIntervals),
		//inNeighborIds(inNeighborIds),
		inNodeTypes(inTypes),
		inNodeIds(inIds),
		inBorderNodeFlags(inFlags),
		inDistMatrix(inMatrix),
		outNodeTypes(outTypes),
		outNodeIds(outIds),
		outBorderNodeFlags(outFlags),
		outDistMatrix(outMatrix),
		outValidSubgraphFlags(outSubgraphFlags)
	{}
	
	//__host__ __device__ FORCE_INLINE void invalidateSubgraph(unsigned int subgraphStartLocation)
	//{
	//	for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
	//	{
	//		outNodeIds[subgraphStartLocation + localNodeId] = graphSize2;
	//		outBorderNodeFlags[subgraphStartLocation + localNodeId] = 0u;
	//	}
	//}

	__host__ __device__	void operator()(const size_t& aId_s)
	{
		unsigned int aId = (unsigned int)aId_s;
		//unsigned int subgraphSeedNodeId = subgraphsPerSeedNode == 0u ? aId : aId / subgraphsPerSeedNode;
		//unsigned int subgraphOffset = subgraphsPerSeedNode == 0u ? 0u : aId % subgraphsPerSeedNode;
		//unsigned int subgraphStartLocation = subgraphOffset * subgraphSize + subgraphSeedNodeId * subgraphsPerSeedNode * subgraphSize;

		unsigned int subgraphStartLocation = aId * subgraphSize;// (aId % 32) * subgraphSize;

		unsigned int interiorNodesCount = 0;
		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
		{
			if (inBorderNodeFlags[subgraphStartLocation + localNodeId] == 0u)
			{
				//only match the nodes on the inner border and border
				outNodeIds[subgraphStartLocation + localNodeId] = graphSize2;
				++interiorNodesCount;
			}
			else
			{
				outNodeIds[subgraphStartLocation + localNodeId] = 0u;
			}
			outBorderNodeFlags[subgraphStartLocation + localNodeId] = 0u;
		}
		
		if (interiorNodesCount == 0 || subgraphSize - interiorNodesCount < 3)
		{
			//invalid subgraph - too few nodes in the cut, or no interior nodes
			//invalidateSubgraph(subgraphStartLocation);
			return;
		}

		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize; ++localNodeId)
		{
			if (inBorderNodeFlags[subgraphStartLocation + localNodeId] == 0u)
				continue;

			unsigned int inNodeId = inNodeIds[subgraphStartLocation + localNodeId];
			unsigned int inNodeType = inNodeTypes[inNodeId];
			bool foundAMatch = false;
			for (unsigned int nodeId2 = outNodeIds[subgraphStartLocation + localNodeId]; nodeId2 < graphSize2 && !foundAMatch; ++nodeId2)
			{
				//unsigned int nodeId2 = (nodeId2_it + 16) % graphSize2;
				bool matches = outNodeTypes[nodeId2] == inNodeType;
				if (!matches)
					continue;
				bool skip = false;
				for (unsigned int recordedNodeId = 0; recordedNodeId < localNodeId && !skip; ++recordedNodeId)
				{
					if (outNodeIds[subgraphStartLocation + recordedNodeId] == nodeId2)
						skip = true;//already participates
					if (inBorderNodeFlags[subgraphStartLocation + recordedNodeId] == 0u)
						continue;//skip interior nodes
					unsigned int pairNodeId = inNodeIds[subgraphStartLocation + recordedNodeId];
					const float targetDist = inDistMatrix[inNodeId + graphSize1 * pairNodeId];
					unsigned int pairNodeId2 = outNodeIds[subgraphStartLocation + recordedNodeId];
					const float currentDist = outDistMatrix[nodeId2 + graphSize2 * pairNodeId2];
					if (fabsf(currentDist - targetDist) > spatialTolerance)
						skip = true;//incompatible with previous participants
				}
				if (skip)
					continue;

				for (unsigned int pairingNodeId = 0; pairingNodeId < subgraphSize; ++pairingNodeId)
				{
					if (inBorderNodeFlags[subgraphStartLocation + pairingNodeId] == 0u)
						continue;
					if (pairingNodeId == localNodeId)
						continue;
					unsigned int pairNodeId = inNodeIds[subgraphStartLocation + pairingNodeId];
					unsigned int pairNodeType = inNodeTypes[pairNodeId];
					const float targetDist = inDistMatrix[inNodeId + graphSize1 * pairNodeId];
					bool foundMatchingPair = false;
					for (unsigned int pairingNodeId2 = 0u; pairingNodeId2 < graphSize2 && !foundMatchingPair; ++pairingNodeId2)
					{
						if (pairingNodeId2 == nodeId2)
							continue;
						if (outNodeTypes[pairingNodeId2] != pairNodeType)
							continue;
						const float currentDist = outDistMatrix[nodeId2 + graphSize2 * pairingNodeId2];
						if (fabsf(currentDist - targetDist) < spatialTolerance)
							foundMatchingPair = true;
					}//end for all other nodes in the second graph
					if (!foundMatchingPair)
						matches = false;
				}//end for all other nodes in the cut
				if (matches)
				{
					outNodeIds[subgraphStartLocation + localNodeId] = nodeId2;
					outBorderNodeFlags[subgraphStartLocation + localNodeId] = inBorderNodeFlags[subgraphStartLocation + localNodeId];
					foundAMatch = true;
				}
			}//end for all nodes in the second graph
			if (!foundAMatch)
			{
				//backtrack
				if (localNodeId > 0) --localNodeId;
				while (inBorderNodeFlags[subgraphStartLocation + localNodeId] == 0u && localNodeId > 0) --localNodeId;

				if (localNodeId <= 1)
				{
					//did not find a mathcing node in the second graph, invalidate the subgraph
					//invalidateSubgraph(subgraphStartLocation);
					return;
				}

				outNodeIds[subgraphStartLocation + localNodeId] = outNodeIds[subgraphStartLocation + localNodeId] + 1;
				outBorderNodeFlags[subgraphStartLocation + localNodeId] = 0u;
				--localNodeId;//cancel out post-increment from the loop
			}
		}//end for all nodes in the cut

		//double check selected matching nodes
		bool foundMismatch = false;
		for (unsigned int localNodeId = 0u; localNodeId < subgraphSize && !foundMismatch; ++localNodeId)
		{
			if (inBorderNodeFlags[subgraphStartLocation + localNodeId] == 0u)
				continue;

			unsigned int inNodeId = inNodeIds[subgraphStartLocation + localNodeId];
			unsigned int inNodeType = inNodeTypes[inNodeId];
			unsigned int outNodeId = outNodeIds[subgraphStartLocation + localNodeId];
			unsigned int outNodeType = outNodeTypes[outNodeId];
			if (inNodeType != outNodeType)
			{
				foundMismatch = true;
				break;
			}

			for (unsigned int pairingNodeId = 0; pairingNodeId < subgraphSize; ++pairingNodeId)
			{
				if (inBorderNodeFlags[subgraphStartLocation + pairingNodeId] == 0u)
					continue;
				if (pairingNodeId == localNodeId)
					continue;
				unsigned int inPairNodeId = inNodeIds[subgraphStartLocation + pairingNodeId];
				unsigned int outPairNodeId = outNodeIds[subgraphStartLocation + pairingNodeId];
				const float targetDist = inDistMatrix[inNodeId + graphSize1 * inPairNodeId];
				const float currentDist = outDistMatrix[outNodeId + graphSize2 * outPairNodeId];
				if (fabsf(currentDist - targetDist) > spatialTolerance)
				{
					foundMismatch = true;
					break;
				}
			}//end for each other node in the cut
		}//end for each node in the cut
		
		//if (foundMismatch)
		//{
		//	invalidateSubgraph(subgraphStartLocation);
		//	return;
		//}

		if(!foundMismatch)
		{
			outValidSubgraphFlags[aId] = 1u;
		}

	}

};

class TransformationEstimator
{
public:
	unsigned int subgraphSize;

	thrust::device_ptr<float3> positions1;
	thrust::device_ptr<float3> positions2;

	thrust::device_ptr<unsigned int> nodeIds1;
	thrust::device_ptr<unsigned int> borderNodeFlags;
	thrust::device_ptr<unsigned int> nodeIds2;

	thrust::device_ptr<unsigned int> outValidSubgraphFlags;

	thrust::device_ptr<float3> outTranslation1;
	thrust::device_ptr<float3> outTranslation2;
	thrust::device_ptr<float> tmpCovMatrix;
	thrust::device_ptr<float> tmpDiagonalW;
	thrust::device_ptr<float> tmpMatrixV;
	thrust::device_ptr<float> tmpVecRV;
	thrust::device_ptr<quaternion4f> outRotation2;


	TransformationEstimator(
		unsigned int aSampleSize,
		thrust::device_ptr<float3> aPositions1,
		thrust::device_ptr<float3> aPositions2,
		thrust::device_ptr<unsigned int> inIds,
		thrust::device_ptr<unsigned int> inFlags,
		thrust::device_ptr<unsigned int> outIds,
		thrust::device_ptr<unsigned int> outSubgraphFlags,
		thrust::device_ptr<float3> aTranslation1,
		thrust::device_ptr<float3> aTranslation2,
		thrust::device_ptr<float> aCovMatrix,
		thrust::device_ptr<float> aDiagonalW,
		thrust::device_ptr<float> aMatrixV,
		thrust::device_ptr<float> aVecRV,
		thrust::device_ptr<quaternion4f> aOutRot
	) : subgraphSize(aSampleSize),
		positions1(aPositions1),
		positions2(aPositions2),
		nodeIds1(inIds),
		borderNodeFlags(inFlags),
		nodeIds2(outIds),
		outValidSubgraphFlags(outSubgraphFlags),
		outTranslation1(aTranslation1),
		outTranslation2(aTranslation2),
		tmpCovMatrix(aCovMatrix),
		tmpDiagonalW(aDiagonalW),
		tmpMatrixV(aMatrixV),
		tmpVecRV(aVecRV),
		outRotation2(aOutRot)
	{}

	__host__ __device__	void operator()(const size_t& aId_s)
	{
		unsigned int aId = (unsigned int)aId_s;
		if (outValidSubgraphFlags[aId] == 0u)
			return;

		unsigned int subgraphStartLocation = aId * subgraphSize;// (aId % 32) * subgraphSize;

		//Compute the means of the border node locations
		float3 center1 = make_float3(0.f, 0.f, 0.f);
		float3 center2 = make_float3(0.f, 0.f, 0.f);
		float numPoints = 0.f;
		for (unsigned int i = 0u; i < subgraphSize; ++i)
		{
			if (borderNodeFlags[subgraphStartLocation + i] != 0u)
			{
				center1 += positions1[nodeIds1[subgraphStartLocation + i]];
				center2 += positions2[nodeIds2[subgraphStartLocation + i]];
				numPoints += 1.f;
			}
		}
		center1 /= numPoints;
		center2 /= numPoints;

		//Compute covariance matrix
		float* covMat = thrust::raw_pointer_cast(tmpCovMatrix + aId * 3 * 3);
		for (unsigned int i = 0u; i < subgraphSize; ++i)
		{
			if (borderNodeFlags[subgraphStartLocation + i] != 0u)
			{
				float3 vec1 = positions1[nodeIds1[subgraphStartLocation + i]] - center1;
				float3 vec2 = positions2[nodeIds2[subgraphStartLocation + i]] - center2;

				covMat[0 * 3 + 0] += vec2.x * vec1.x;
				covMat[1 * 3 + 0] += vec2.y * vec1.x;
				covMat[2 * 3 + 0] += vec2.z * vec1.x;

				covMat[0 * 3 + 1] += vec2.x * vec1.y;
				covMat[1 * 3 + 1] += vec2.y * vec1.y;
				covMat[2 * 3 + 1] += vec2.z * vec1.y;

				covMat[0 * 3 + 2] += vec2.x * vec1.z;
				covMat[1 * 3 + 2] += vec2.y * vec1.z;
				covMat[2 * 3 + 2] += vec2.z * vec1.z;
			}
		}
		//Singular Value Decomposition
		float* diag = thrust::raw_pointer_cast(tmpDiagonalW + aId * 3);
		float* vMat = thrust::raw_pointer_cast(tmpMatrixV + aId * 3 * 3);
		float* tmp = thrust::raw_pointer_cast(tmpVecRV + aId * 3);

		svd::svdcmp(covMat, 3, 3, diag, vMat, tmp);

		//Rotation is V * transpose(U)		
		for (unsigned int row = 0; row < 3; ++row)
		{
			for (unsigned int col = 0; col < 3; ++col)
			{
				tmp[col] =
					vMat[row * 3 + 0] * covMat[col * 3 + 0] +
					vMat[row * 3 + 1] * covMat[col * 3 + 1] +
					vMat[row * 3 + 2] * covMat[col * 3 + 2];
			}
			vMat[row * 3 + 0] = tmp[0];
			vMat[row * 3 + 1] = tmp[1];
			vMat[row * 3 + 2] = tmp[2];
		}


		float rotDet = determinant(
			vMat[0], vMat[3], vMat[6],
			vMat[1], vMat[4], vMat[7],
			vMat[2], vMat[5], vMat[8]
		);

		if (rotDet < 0.f)
		{
			vMat[6] = -vMat[6];
			vMat[7] = -vMat[7];
			vMat[8] = -vMat[8];
			rotDet = -rotDet;
		}

		if (fabsf(rotDet - 1.f)> EPS)
			outValidSubgraphFlags[aId] = 0u;


		quaternion4f rotation(
			vMat[0], vMat[3], vMat[6],
			vMat[1], vMat[4], vMat[7],
			vMat[2], vMat[5], vMat[8]
		);
		outTranslation1[aId] = center1;
		outTranslation2[aId] = center2;
		outRotation2[aId] = rotation;
	}

};


__host__ std::string VariationGenerator::operator()(const char * aFilePath1, const char * aFilePath2,
	WFObject & aObj1, WFObject & aObj2, Graph & aGraph1, Graph & aGraph2, float aRelativeThreshold)
{
	cudastd::timer timer;
	cudastd::timer intermTimer;

	thrust::host_vector<float3> objCenters1;
	thrust::host_vector<float> objSizes1;

	ObjectCenterExporter()(aObj1, objCenters1, objSizes1, 0.3333f);

	thrust::host_vector<float3> objCenters2;
	thrust::host_vector<float> objSizes2;

	ObjectCenterExporter()(aObj2, objCenters2, objSizes2, 0.3333f);

	thrust::device_vector<float3> centersDevice1(objCenters1);
	thrust::device_vector<float> pairwiseDistMatrix1(objCenters1.size() * objCenters1.size());
	thrust::counting_iterator<size_t> first(0u);
	thrust::counting_iterator<size_t> last1(pairwiseDistMatrix1.size());
	DistanceMatrixWriter writeDistances1(objCenters1.size(), pairwiseDistMatrix1.data(), centersDevice1.data());
	thrust::for_each(first, last1, writeDistances1);

	thrust::host_vector<unsigned int> nodeTypes1Host(aGraph1.numNodes(), (unsigned int)aObj1.materials.size());
	for (size_t nodeId = 0; nodeId < aObj1.objects.size(); ++nodeId)
	{
		size_t faceId = aObj1.objects[nodeId].x;
		size_t materialId = aObj1.faces[faceId].material;
		nodeTypes1Host[nodeId] = (unsigned int)materialId;
	}
	thrust::device_vector<unsigned int> nodeTypes1(nodeTypes1Host);

	thrust::device_vector<float3> centersDevice2(objCenters2);
	thrust::device_vector<float> pairwiseDistMatrix2(objCenters2.size() * objCenters2.size());
	thrust::counting_iterator<size_t> last2(pairwiseDistMatrix2.size());
	DistanceMatrixWriter writeDistances2(objCenters2.size(), pairwiseDistMatrix2.data(), centersDevice2.data());
	thrust::for_each(first, last2, writeDistances2);

	thrust::host_vector<unsigned int> nodeTypes2Host(aGraph2.numNodes(), (unsigned int)aObj2.materials.size());
	for (size_t nodeId = 0u; nodeId < aObj2.objects.size(); ++nodeId)
	{
		size_t faceId = aObj2.objects[nodeId].x;
		size_t materialId = aObj2.faces[faceId].material;
		nodeTypes2Host[nodeId] = (unsigned int)materialId;
	}
	thrust::device_vector<unsigned int> nodeTypes2(nodeTypes2Host);

	initTime = intermTimer.get();
	intermTimer.start();

	const unsigned int numSubgraphSamples = 100u * (unsigned int)objCenters1.size();
	const unsigned int subgraphSampleSize = (unsigned int)objCenters1.size() / 2u;

	if (subgraphSampleSize < 3)
		return "";

	thrust::device_vector<unsigned int> subgraphNodeIds1(numSubgraphSamples * subgraphSampleSize);
	thrust::device_vector<unsigned int> subgraphBorderFlags1(numSubgraphSamples * subgraphSampleSize);

	thrust::device_vector<unsigned int> subgraphNodeIds2(numSubgraphSamples * subgraphSampleSize);
	thrust::device_vector<unsigned int> subgraphBorderFlags2(numSubgraphSamples * subgraphSampleSize);

	SubgraphInitializer initSubgraphSamples(
		(unsigned int)objCenters1.size(),
		subgraphSampleSize,
		numSubgraphSamples,
		aGraph1.intervals.data(),
		aGraph1.adjacencyVals.data(),
		subgraphNodeIds1.data(),
		subgraphBorderFlags1.data());

	thrust::counting_iterator<size_t> lastSubgraph(numSubgraphSamples);
	thrust::for_each(first, lastSubgraph, initSubgraphSamples);

	samplingTime = intermTimer.get();
	intermTimer.start();

//#ifdef _DEBUG
//	outputDeviceVector("Subgraph node ids     1: ", subgraphNodeIds1);
//	outputDeviceVector("Subgraph border flags 1: ", subgraphBorderFlags1);
//#endif

	///////////////////////////////////////////////////////////////////////////////////
	//Find matching cuts in both sub-graphs

	float3 minBound, maxBound;
	ObjectBoundsExporter()(aObj1, minBound, maxBound);
	const float boundsDiagonal = len(maxBound - minBound);
	const float spatialTolerance = boundsDiagonal * 0.577350269f * aRelativeThreshold;

	thrust::device_vector<unsigned int> validSubgraphFlags(numSubgraphSamples, 0u);

	CutMatching matchCuts(
		(unsigned int)objCenters1.size(),
		(unsigned int)objCenters2.size(),
		subgraphSampleSize,
		numSubgraphSamples,
		spatialTolerance,
		nodeTypes1.data(),
		subgraphNodeIds1.data(),
		subgraphBorderFlags1.data(),
		pairwiseDistMatrix1.data(),
		nodeTypes2.data(),
		subgraphNodeIds2.data(),
		subgraphBorderFlags2.data(),
		pairwiseDistMatrix2.data(),
		validSubgraphFlags.data()
	);

	//thrust::counting_iterator<size_t> lastSubgraphDbg(4);
	thrust::for_each(first, lastSubgraph, matchCuts);

	matchingTime = intermTimer.get();
	intermTimer.start();

//#ifdef _DEBUG
//	outputDeviceVector("Subgraph node ids     2: ", subgraphNodeIds2);
//	outputDeviceVector("Subgraph border flags 2: ", subgraphBorderFlags2);
//	outputDeviceVector("Valid subgraph flags   : ", validSubgraphFlags);
//#endif


	///////////////////////////////////////////////////////////////////////////////////
	//Find correspondence transformation between both sub-graphs
	thrust::device_vector<float3> outTranslation1(numSubgraphSamples);
	thrust::device_vector<float3> outTranslation2(numSubgraphSamples);
	thrust::device_vector<float> tmpCovMatrix(numSubgraphSamples * 3 * 3, 0.f);
	thrust::device_vector<float> tmpDiagonalW(numSubgraphSamples * 3);
	thrust::device_vector<float> tmpMatrixV(numSubgraphSamples * 3 * 3);
	thrust::device_vector<float> tmpVecRV(numSubgraphSamples * 3);
	thrust::device_vector<quaternion4f> outRotation2(numSubgraphSamples);

	TransformationEstimator estimateT(
		subgraphSampleSize,
		centersDevice1.data(),
		centersDevice2.data(),
		subgraphNodeIds1.data(),
		subgraphBorderFlags1.data(),
		subgraphNodeIds2.data(),
		validSubgraphFlags.data(),
		outTranslation1.data(),
		outTranslation2.data(),
		tmpCovMatrix.data(),
		tmpDiagonalW.data(),
		tmpMatrixV.data(),
		tmpVecRV.data(),
		outRotation2.data()
	);

	thrust::for_each(first, lastSubgraph, estimateT);

	svdTime = intermTimer.get();
	intermTimer.start();

	///////////////////////////////////////////////////////////////////////////////////
	//Copy back to host
	thrust::host_vector<unsigned int> subgraphNodeIdsHost1(subgraphNodeIds1);
	thrust::host_vector<unsigned int> subgraphBorderFlagsHost1(subgraphBorderFlags1);

	thrust::host_vector<unsigned int> subgraphNodeIdsHost2(subgraphNodeIds2);
	thrust::host_vector<unsigned int> subgraphBorderFlagsHost2(subgraphBorderFlags2);

	thrust::host_vector<unsigned int> graph2Intervals(aGraph2.intervals);
	thrust::host_vector<unsigned int> graph2NbrIds(aGraph2.adjacencyVals);

	thrust::host_vector<unsigned int> validSubgraphFlagsHost(validSubgraphFlags);

	thrust::host_vector<float3> outTranslation1Host(outTranslation1);
	thrust::host_vector<float3> outTranslation2Host(outTranslation2);
	thrust::host_vector<quaternion4f> outRotation2Host(outRotation2);

	unsigned int graphSize1 = (unsigned int)objCenters1.size();
	unsigned int graphSize2 = (unsigned int)objCenters2.size();

	std::string result = "";
	GraphToStringConverter convertToStr;
	CollisionGraphExporter graphExporter;
	numVariations = 0u;
	std::vector<NodeTypeHistogram> variatioHistograms;
	variatioHistograms.push_back(NodeTypeHistogram(nodeTypes1));
	variatioHistograms.push_back(NodeTypeHistogram(nodeTypes2));
	
	cpyBackTime = intermTimer.get();
	intermTimer.start();

	histTime = transformTime = collisionTime = exportTime = conversionTime = 0.f;
	histoChecks = 0u;

	for (unsigned int subgraphId = 0u; subgraphId < numSubgraphSamples; ++subgraphId)
	{
		if (validSubgraphFlagsHost[subgraphId] != 1u)
			continue;

		thrust::host_vector<unsigned int> completeSubgraphFlags2(graphSize2, 0u);
		std::vector<unsigned int> nodeStack;
		unsigned int subgraph2Size = 0u;
		unsigned int complementSize = 0u;
		thrust::host_vector<unsigned int>::iterator subgraphNodeIdsHost1Begin = subgraphNodeIdsHost1.begin() + subgraphId * subgraphSampleSize;
		thrust::host_vector<unsigned int>::iterator subgraphNodeIdsHost2Begin = subgraphNodeIdsHost2.begin() + subgraphId * subgraphSampleSize;
		thrust::host_vector<unsigned int>::iterator subgraphBorderFlagsHost1Begin = subgraphBorderFlagsHost1.begin() + subgraphId * subgraphSampleSize;
		thrust::host_vector<unsigned int>::iterator subgraphBorderFlagsHost2Begin = subgraphBorderFlagsHost2.begin() + subgraphId * subgraphSampleSize;

		//initialize flags at graph cut - 2 -> outside node, 1 -> border node
		for (unsigned int i = 0u; i < subgraphSampleSize; ++i)
		{
			if (*(subgraphBorderFlagsHost2Begin + i) != 0u)
				completeSubgraphFlags2[*(subgraphNodeIdsHost2Begin + i)] = *(subgraphBorderFlagsHost2Begin + i);
			if (*(subgraphBorderFlagsHost2Begin + i) == 1u)
			{
				++subgraph2Size;
				nodeStack.push_back(*(subgraphNodeIdsHost2Begin + i));
			}
			if (*(subgraphBorderFlagsHost2Begin + i) == 2u)
				++complementSize;
		}
		//region grow from each border node
		while (!nodeStack.empty())
		{
			unsigned int nodeId = nodeStack.back();
			nodeStack.pop_back();
			for (unsigned int nbr = graph2Intervals[nodeId]; nbr < graph2Intervals[nodeId + 1]; ++nbr)
			{
				unsigned int nbrId = graph2NbrIds[nbr];
				if (completeSubgraphFlags2[nbrId] == 2u || completeSubgraphFlags2[nbrId] == 1u)
					continue;
				completeSubgraphFlags2[nbrId] = 1u;
				nodeStack.push_back(nbrId);
				++subgraph2Size;
			}
		}
		//check validity
		if (subgraph2Size + complementSize >= graphSize2)
			continue; //should not happen
		unsigned int subgraph1Size = 0u;
		thrust::host_vector<unsigned int> completeSubgraphFlags1(graphSize1, 0u);
		for (unsigned int i = 0u; i < subgraphSampleSize; ++i)
		{
			if (*(subgraphBorderFlagsHost1Begin + i) == 0u || *(subgraphBorderFlagsHost1Begin + i) == 2u)
			{
				completeSubgraphFlags1[*(subgraphNodeIdsHost1Begin + i)] = 1u;
				++subgraph1Size;
			}
		}
		
		intermTimer.start();
		///////////////////////////////////////////////////////////////////////////////////
		//discard variations with repeating node type histograms
		NodeTypeHistogram typeHist(aObj1.materials.size());
		for (auto inTypeIt1 = nodeTypes1Host.begin(); inTypeIt1 != nodeTypes1Host.end(); ++inTypeIt1)
		{
			if (completeSubgraphFlags1[inTypeIt1 - nodeTypes1Host.begin()] == 1u)
			{
				typeHist.typeCounts[*inTypeIt1]++;			
			}
		}
		for (auto inTypeIt2 = nodeTypes2Host.begin(); inTypeIt2 != nodeTypes2Host.end(); ++inTypeIt2)
		{
			if (completeSubgraphFlags2[inTypeIt2 - nodeTypes2Host.begin()] == 1u)
			{
				typeHist.typeCounts[*inTypeIt2]++;
			}
		}

		bool repeatedHistogram = false;
		for (size_t hid = 0u; hid < variatioHistograms.size() && !repeatedHistogram; ++hid)
		{
			++histoChecks;
			if (typeHist == variatioHistograms[hid])
				repeatedHistogram = true;
		}


		histTime += intermTimer.get();
		intermTimer.start();

		if (repeatedHistogram)
			continue;
		variatioHistograms.push_back(typeHist);
		////////////////////////////////////////////////////////////////////////////////////////

		for (unsigned int i = 0u; i < graphSize2; ++i)
		{
			if (completeSubgraphFlags2[i] == 2u)
				completeSubgraphFlags2[i] = 0u;
		}

		//graphExporter.exportSubGraph(aFilePath1, aObj1, aGraph1, numVariations, completeSubgraphFlags1);
		//graphExporter.exportSubGraph(aFilePath2, aObj2, aGraph2, numVariations, completeSubgraphFlags2);

		++numVariations;

		///////////////////////////////////////////////////////////////////////////////////
		//Create the variation by merging the subsets of aObj1 and aObj2
		float3 translation1 = outTranslation1Host[subgraphId];
		float3 translation2 = outTranslation2Host[subgraphId];
		quaternion4f rotation2 = outRotation2Host[subgraphId];
		WFObject variation = WFObjectMerger()(aObj1, translation1, aObj2, translation2, rotation2, completeSubgraphFlags1, completeSubgraphFlags2);
		///////////////////////////////////////////////////////////////////////////////////
		transformTime  += intermTimer.get();
		intermTimer.start();
		///////////////////////////////////////////////////////////////////////////////////
		//Compute the collision graph for the variation
		CollisionDetector detector;
		Graph variationGraph = detector.computeCollisionGraph(variation, 0.01f);
		///////////////////////////////////////////////////////////////////////////////////
		collisionTime += intermTimer.get();
		intermTimer.start();
		///////////////////////////////////////////////////////////////////////////////////
		//Check that the variation graph is valid

		//TODO:Check that the variation graph is valid

		///////////////////////////////////////////////////////////////////////////////////

		std::string fileName1(aFilePath1);
		if (fileName1.find_last_of("/\\") == std::string::npos)
			fileName1 = fileName1.substr(0, fileName1.size() - 5);
		else
			fileName1 = fileName1.substr(fileName1.find_last_of("/\\") + 1, fileName1.size() - fileName1.find_last_of("/\\") - 5);

		std::string fileName2(aFilePath1);
		if (fileName2.find_last_of("/\\") == std::string::npos)
			fileName2 = fileName2.substr(0, fileName2.size() - 5);
		else
			fileName2 = fileName2.substr(fileName2.find_last_of("/\\") + 1, fileName2.size() - fileName2.find_last_of("/\\") - 5);


		std::string objDir = getDirName(aFilePath2);
		std::string variationFilePath = objDir + fileName1 + "_" + fileName2 + "_var_" + itoa((int)numVariations) + ".obj";

		graphExporter.exportCollisionGraph(variationFilePath.c_str(), variation, variationGraph);
		
		exportTime = intermTimer.get();		
		intermTimer.start();

		std::string variationStrings = convertToStr(variation, variationGraph);
		result.append(variationStrings);

		conversionTime += intermTimer.get();
	}

	totalTime = timer.get();

	intermTimer.cleanup();
	timer.cleanup();

	return result;
}


__host__ void VariationGenerator::stats()
{
	std::cerr << "Created "<< numVariations <<" variations in " << totalTime << "ms\n";
	std::cerr << "Initialization in      " << initTime << "ms\n";
	std::cerr << "Subgraph sampling in   " << samplingTime << "ms\n";
	std::cerr << "Graph cut matching in  " << matchingTime << "ms\n";
	std::cerr << "SVD in                 " << svdTime << "ms\n";
	std::cerr << "Mem transfer in        " << cpyBackTime << "ms\n";
	std::cerr << "Histogram check  in    " << histTime << "ms (checked   " << histoChecks << " candidates)\n";
	std::cerr << "Obj transformation in  " << transformTime << "ms\n";
	std::cerr << "Collision detection in " << collisionTime << "ms\n";
	std::cerr << "File export in         " << exportTime << "ms\n";
	std::cerr << "String conversion      " << conversionTime << "ms\n";
}