#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#include "WFObjectToString.h"
#include "UniformGridSortBuilderTest.h"
#include "GraphTest.h"
#include "CollisionTest.h"
#include "ShapeVariationTest.h"
#include "RNGTest.h"

#include <thrust/detail/config.h>

int main()
{

#if 0 //THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		std::cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n";
		return 1;
	}
#endif
	

	RNGTest rngTest;
	int rng_test_result = rngTest.testAll();
	//graphTest.testAll(1000);

	if (rng_test_result != 0)
	{
		std::cerr << "Random number generator test failed!\n";
		return rng_test_result;
	}
	else
	{
		std::cerr << "Random number generator test passed.\n";
	}

	UniformGridSortBuildTest uniformGridTest;
	int ugrid_test_result = uniformGridTest.testAll("../scenes/church/church.obj", 32, 16, 24);
	
	if (ugrid_test_result != 0)
	{
		std::cerr << "Uniform grid construction test failed!\n";
		return ugrid_test_result;
	}
	else
	{
		std::cerr << "Uniform grid construction test passed.\n";

	}
	
	GraphTest graphTest;
	int graph_test_result = graphTest.testAll(1000);
	//graphTest.testAll(1000);
	
	if (graph_test_result != 0)
	{
		std::cerr << "Graph construction test failed!\n";
		return graph_test_result;
	}
	else
	{
		std::cerr << "Graph construction test passed.\n";
	}
	
	CollisionTest collTest;
	int coll_test_result = collTest.testAll("../scenes/castle/castle.obj");
	if (coll_test_result != 0)
	{
		std::cerr << "Collision detection test failed!\n";
		return coll_test_result;
	}
	else
	{
		std::cerr << "Collision detection test passed. \n";
	}
	
	std::cerr << "---------------------------------------------------------------------\n";
	const char* obj2strTestFile = "../scenes/church/church.obj";
	std::cerr << obj2strTestFile << " converted to \n"
		<< WFObjectToString(obj2strTestFile) << "\n";


	const char* variationFile1 = "../scenes/church/test/variant_c19.obj";
	const char* variationFile2 = "../scenes/church/test/variant_c28.obj";
	//const char* variationFile1 = "../scenes/skyscraper/test/variant_100.obj";
	//const char* variationFile2 = "../scenes/skyscraper/test/variant_101.obj";
	//const char* variationFile1 = "../scenes/playground/variant_000.obj";
	//const char* variationFile2 = "../scenes/playground/variant_001.obj";

	ShapeVariationTest variationTest;
	int var_test_result = variationTest.testAll(variationFile1, variationFile2);
	if (var_test_result != 0)
	{
		std::cerr << "Shape variation test failed!\n";
		return var_test_result;
	}
	else
	{
		std::cerr << "Shape variation test passed. \n";
	}




#if 0// THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
     //hipDeviceReset must be called before exiting in order for profiling and
     //tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
		std::cerr << "hipDeviceReset failed!\n";
        return 1;
    }
#endif

    return 0;
}

