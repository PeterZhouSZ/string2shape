
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "UniformGridSortBuilderTest.h"
#include "GraphTest.h"
#include "CollisionTest.h"


int main()
{
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}
	CollisionTest collTest;
	collTest.testAll("../scenes/castle.obj");

	UniformGridSortBuildTest uniformGridTest;
	uniformGridTest.testAll("../scenes/castle.obj", 32, 32, 32);

	GraphTest graphTest;
	graphTest.testAll(100);
	//graphTest.testAll(1000);



    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

