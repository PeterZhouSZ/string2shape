
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "UniformGridSortBuilderTest.h"
#include "GraphTest.h"
#include "CollisionTest.h"


int main()
{
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	UniformGridSortBuildTest uniformGridTest;
	int ugrid_test_result = uniformGridTest.testAll("../scenes/castle/castle.obj", 32, 16, 24);
	if (ugrid_test_result != 0)
	{
		fprintf(stderr, "Uniform grid construction test failed!\n");
		return ugrid_test_result;
	}
	else
	{
		fprintf(stderr, "Uniform grid construction test passed.\n");
	}

	GraphTest graphTest;
	int graph_test_result = graphTest.testAll(100);
	//graphTest.testAll(1000);

	if (graph_test_result != 0)
	{
		fprintf(stderr, "Graph construction test failed!\n");
		return graph_test_result;
	}
	else
	{
		fprintf(stderr, "Graph construction test passed.\n");
	}

	CollisionTest collTest;
	collTest.testAll("../scenes/castle/castle.obj");
	fprintf(stderr, "Collision detection test passed. (check collision graph)\n");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

