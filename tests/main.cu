
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "UniformGridSortBuilderTest.h"


int main()
{
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	UniformGridSortBuildTest uniformGridTest;
	uniformGridTest.testAll("../scenes/castle.obj", 32, 32, 32);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

